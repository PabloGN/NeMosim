#include "hip/hip_runtime.h"
#ifndef TARGET_PARTITIONS_CU
#define TARGET_PARTITIONS_CU

#include "outgoing.cu_h"

__constant__ size_t c_outgoingPitch; // word pitch


__host__
outgoing_t
make_outgoing(pidx_t partition, delay_t delay, uint warps)
{
	assert(partition < MAX_PARTITION_COUNT);
	assert(delay < MAX_DELAY);
	assert(warps < MAX_SYNAPSE_WARPS);
	assert(MAX_PARTITION_COUNT < 256);
	assert(MAX_DELAY < 256);
	assert(MAX_SYNAPSE_WARPS <= 256);
	return make_uchar4((uchar) partition, (uchar) delay, (uchar) warps, 0);
}



__host__
bool
operator<(const outgoing_t& a, const outgoing_t& b)
{
	return a.x < b.x || (a.x == b.x && a.y < b.y);
}



__host__
void
setOutgoingPitch(size_t targetPitch)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_outgoingPitch),
				&targetPitch, sizeof(size_t), 0, hipMemcpyHostToDevice));
}



__host__ __device__
size_t
outgoingRow(pidx_t partition, nidx_t neuron, size_t pitch)
{
	//! \todo factor out addressing function and share with the 'counts' function
	return (partition * MAX_PARTITION_SIZE + neuron) * pitch;
}



__device__
uint
outgoingTargetPartition(outgoing_t out)
{
	return (uint) out.x;
}



__device__
uint
outgoingDelay(outgoing_t out)
{
	return (uint) out.y;
}



__device__
uint
outgoingWarps(outgoing_t out)
{
	return (uint) out.z;
}



__device__
outgoing_t
outgoing(uint presynaptic,
		uint jobIdx,
		outgoing_t* g_targets)
{
	size_t addr = outgoingRow(CURRENT_PARTITION, presynaptic, c_outgoingPitch);
	return g_targets[addr + jobIdx];
}



/*! \return
 *		the number of jobs for a particular firing neuron in the current
 *		partition */
__device__
uint
outgoingCount(uint presynaptic, uint* g_counts)
{
	return g_counts[CURRENT_PARTITION * MAX_PARTITION_SIZE + presynaptic];
}


#endif
