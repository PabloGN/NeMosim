#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
// Kernel configuration
//-----------------------------------------------------------------------------
//
// Different clusters may have slightly different configuration, e.g. with
// respect to what external inputs they require. This information is all kept
// in constant memory and is thus cached. 
// 
// It is not possible to dynamically size data structures in constant memory,
// so we simply set some upper limit on the number of thread blocks
// (MAX_THREAD_BLOCKS) and size the data structures statically.
//-----------------------------------------------------------------------------


#include <cutil.h>
#include "kernel.cu_h"


/* Network-wide configuration */

#define NPARAM_maxPartitionSize 0
#define NPARAM_maxDelay 1
#define NPARAM_pitch32 2
#define NPARAM_pitchL0 3
#define NPARAM_sizeL0 4
#define NPARAM_pitchL1 5
#define NPARAM_sizeL1 6
// STDP parameters
#define NPARAM_rpitchL0 7
#define NPARAM_rsizeL0 8
#define NPARAM_rpitchL1 9
#define NPARAM_rsizeL1 10
#define NPARAM_COUNT 11 

/* Configuration array is stored in constant memory, and is loaded in
 * (parallel) into shared memory for each thread block */
__constant__ uint c_networkParameters[NPARAM_COUNT];
__shared__ uint s_networkParameters[NPARAM_COUNT];

/* Some more pleasant names for the parameters */
//! \todo auto-generate
#define s_maxPartitionSize s_networkParameters[NPARAM_maxPartitionSize]
#define s_maxDelay         s_networkParameters[NPARAM_maxDelay]
#define s_pitch32          s_networkParameters[NPARAM_pitch32]
#define s_pitchL0          s_networkParameters[NPARAM_pitchL0]
#define s_sizeL0           s_networkParameters[NPARAM_sizeL0]
#define s_pitchL1          s_networkParameters[NPARAM_pitchL1]
#define s_sizeL1           s_networkParameters[NPARAM_sizeL1]
#define s_rpitchL0         s_networkParameters[NPARAM_rpitchL0]
#define s_rsizeL0          s_networkParameters[NPARAM_rsizeL0]
#define s_rpitchL1         s_networkParameters[NPARAM_rpitchL1]
#define s_rsizeL1          s_networkParameters[NPARAM_rsizeL1]


#define SET_CONSTANT(symbol, val) param[NPARAM_ ## symbol] = val

__host__
void
configureKernel(RTDATA rtdata)
{
	std::vector<uint> param(NPARAM_COUNT);
	SET_CONSTANT(maxPartitionSize, rtdata->maxPartitionSize);
	SET_CONSTANT(maxDelay, rtdata->maxDelay());
	SET_CONSTANT(pitch32,  rtdata->pitch32());
	SET_CONSTANT(pitchL0,  rtdata->cm(CM_L0)->df_pitch());
	SET_CONSTANT(sizeL0,   rtdata->cm(CM_L0)->df_planeSize());
	SET_CONSTANT(pitchL1,  rtdata->cm(CM_L1)->df_pitch());
	SET_CONSTANT(sizeL1,   rtdata->cm(CM_L1)->df_planeSize());
	SET_CONSTANT(rpitchL0, rtdata->cm(CM_L0)->dr_pitch());
	SET_CONSTANT(rsizeL0,  rtdata->cm(CM_L0)->dr_planeSize());
	SET_CONSTANT(rpitchL1, rtdata->cm(CM_L1)->dr_pitch());
	SET_CONSTANT(rsizeL1,  rtdata->cm(CM_L1)->dr_planeSize());
	CUDA_SAFE_CALL(
			hipMemcpyToSymbol(HIP_SYMBOL(c_networkParameters),
				&param[0], 
				sizeof(uint)*NPARAM_COUNT, 
				0,
				hipMemcpyHostToDevice));
}


#define LOAD_CONSTANT(symbol) s_ ## symbol = c_ ## symbol

__device__
void
loadNetworkParameters()
{
	if(threadIdx.x < NPARAM_COUNT) {
		s_networkParameters[threadIdx.x] = c_networkParameters[threadIdx.x];
	}
	__syncthreads();
}


/* Per-partition configuration */

__constant__ uint c_maxL0SynapsesPerDelay    [MAX_THREAD_BLOCKS];
__constant__ uint c_maxL0RevSynapsesPerDelay [MAX_THREAD_BLOCKS];
__constant__ uint c_maxL1SynapsesPerDelay    [MAX_THREAD_BLOCKS];
__constant__ uint c_maxL1RevSynapsesPerDelay [MAX_THREAD_BLOCKS];

template<class T>
__host__
void
configurePartition(const T& symbol, const std::vector<uint>& values)
{
	std::vector<int> buf(MAX_THREAD_BLOCKS, 0);
	std::copy(values.begin(), values.end(), buf.begin());
	CUDA_SAFE_CALL(
		hipMemcpyToSymbol(HIP_SYMBOL(
			symbol), &buf[0],
			MAX_THREAD_BLOCKS*sizeof(uint),
			0, hipMemcpyHostToDevice));
}


__constant__ uint c_partitionSize[MAX_THREAD_BLOCKS];

__host__
void
configurePartitionSize(size_t n, const uint* d_partitionSize)
{
	CUDA_SAFE_CALL(
		hipMemcpyToSymbol(HIP_SYMBOL(
			c_partitionSize), d_partitionSize,
			MAX_THREAD_BLOCKS*sizeof(uint), 
			0, hipMemcpyHostToDevice));
}
