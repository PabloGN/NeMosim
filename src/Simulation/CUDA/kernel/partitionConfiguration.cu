#include "hip/hip_runtime.h"
#ifndef PARTITION_CONFIGURATION_CU
#define PARTITION_CONFIGURATION_CU

//-----------------------------------------------------------------------------
// Kernel configuration
//-----------------------------------------------------------------------------
//
// Different clusters may have slightly different configuration, e.g. with
// respect to what external inputs they require. This information is all kept
// in constant memory and is thus cached. 
// 
// It is not possible to dynamically size data structures in constant memory,
// so we simply set some upper limit on the number of thread blocks
// (MAX_THREAD_BLOCKS) and size the data structures statically.
//-----------------------------------------------------------------------------


#include "util.h"
#include "kernel.cu_h"
#include "kernel.h"
#include "RuntimeData.hpp"
#include "ConnectivityMatrix.hpp"


/* Network-wide configuration */

#define NPARAM_maxPartitionSize 0
#define NPARAM_maxDelay         1
#define NPARAM_pitch32          2
#define NPARAM_f1_pitch         5
#define NPARAM_f1_size          6
#define NPARAM_pitch64          7
#define NPARAM_COUNT            8

/* Configuration array is stored in constant memory, and is loaded in
 * (parallel) into shared memory for each thread block */
__constant__ uint c_networkParameters[NPARAM_COUNT];
__shared__ uint s_networkParameters[NPARAM_COUNT];

/* Some more pleasant names for the parameters */
//! \todo auto-generate
#define s_maxPartitionSize s_networkParameters[NPARAM_maxPartitionSize]
#define s_maxDelay         s_networkParameters[NPARAM_maxDelay]
#define s_pitch32          s_networkParameters[NPARAM_pitch32]
#define s_pitch64          s_networkParameters[NPARAM_pitch64]
#define sf1_pitch          s_networkParameters[NPARAM_f1_pitch]
#define sf1_size           s_networkParameters[NPARAM_f1_size]


#define SET_CONSTANT(symbol, val) param[NPARAM_ ## symbol] = val

__host__
void
configureKernel(RTDATA rtdata)
{
	std::vector<uint> param(NPARAM_COUNT);
	SET_CONSTANT(maxPartitionSize, rtdata->maxPartitionSize);
	SET_CONSTANT(maxDelay,  rtdata->maxDelay());
	SET_CONSTANT(pitch32,   rtdata->pitch32());
	SET_CONSTANT(pitch64,   rtdata->pitch64());
	//! \todo can remove df_pitch and df_planeSize public methods when removing old L1 FCM
	SET_CONSTANT(f1_pitch,  rtdata->cm(CM_L1)->df_pitch());
	SET_CONSTANT(f1_size,   rtdata->cm(CM_L1)->df_planeSize());
	CUDA_SAFE_CALL(
			hipMemcpyToSymbol(HIP_SYMBOL(c_networkParameters),
				&param[0], 
				sizeof(uint)*NPARAM_COUNT, 
				0,
				hipMemcpyHostToDevice));
}


__device__
void
loadNetworkParameters()
{
	if(threadIdx.x < NPARAM_COUNT) {
		s_networkParameters[threadIdx.x] = c_networkParameters[threadIdx.x];
	}
	__syncthreads();
}


/* Per-partition configuration */

__constant__ uint cf1_maxSynapsesPerDelay[MAX_THREAD_BLOCKS];

template<class T>
__host__
void
configurePartition(const T& symbol, const std::vector<uint>& values)
{
	std::vector<int> buf(MAX_THREAD_BLOCKS, 0);
	std::copy(values.begin(), values.end(), buf.begin());
	CUDA_SAFE_CALL(
		hipMemcpyToSymbol(HIP_SYMBOL(
			symbol), &buf[0],
			MAX_THREAD_BLOCKS*sizeof(uint),
			0, hipMemcpyHostToDevice));
}


__constant__ uint c_partitionSize[MAX_THREAD_BLOCKS];

__host__
void
configurePartitionSize(size_t n, const uint* d_partitionSize)
{
	CUDA_SAFE_CALL(
		hipMemcpyToSymbol(HIP_SYMBOL(
			c_partitionSize), d_partitionSize,
			MAX_THREAD_BLOCKS*sizeof(uint), 
			0, hipMemcpyHostToDevice));
}

#endif
