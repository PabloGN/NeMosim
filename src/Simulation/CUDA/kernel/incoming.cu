#include "hip/hip_runtime.h"
#ifndef L1_SPIKE_BUFFER_CU
#define L1_SPIKE_BUFFER_CU

#include "kernel.cu_h"
#include "incoming.cu_h"


__constant__ size_t c_incomingPitch; // word pitch


__host__
void
setIncomingPitch(size_t pitch)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_incomingPitch),
				&pitch, sizeof(size_t), 0, hipMemcpyHostToDevice));
}



/*! \return the buffer number to use for the given delay, given current cycle */
__device__
uint
incomingSlot(uint cycle, uint delay1)
{
	return (cycle + delay1) % MAX_DELAY;
}



/* Return offset into full buffer data structure to beginning of buffer for a
 * particular targetPartition and a particular delay. */
__device__
uint
incomingBufferStart(uint targetPartition, uint cycle, uint delay1)
{
	return (targetPartition * MAX_DELAY + incomingSlot(cycle, delay1)) * c_incomingPitch;
}



__device__
incoming_t
getIncoming(uint cycle, uint offset, incoming_t* g_incoming)
{
	return g_incoming[incomingBufferStart(CURRENT_PARTITION, cycle, 0) + offset];
}




/*! \return incoming spike group from a particular source */
__device__
incoming_t
make_incoming(uint sourcePartition, uint sourceNeuron, uint delay)
{
	ASSERT(sourcePartition < (1<<8));
	ASSERT(sourceNeuron < (1<<16));
	ASSERT(delay < (1<<8));
	return make_uchar4(
			uchar(sourcePartition),
			uchar(sourceNeuron >> 8),   // MSB
			uchar(sourceNeuron & 0xff), // LSB
			uchar(delay));
}


__device__
uint
incomingDelay(incoming_t in)
{
	return (uint) in.w;
}


__device__
uint
incomingPartition(incoming_t in)
{
	return (uint) in.x;
}



__device__
uint
incomingNeuron(incoming_t in)
{
	return (((uint) in.y) << 8) | ((uint) in.z);
}



/*! \return address into matrix with number of incoming synapse groups */
__device__
size_t
incomingCountAddr(uint targetPartition, uint cycle, uint delay1)
{
	return targetPartition * MAX_DELAY + incomingSlot(cycle, delay1);
}


#endif
