#ifndef L1_SPIKE_BUFFER_CU
#define L1_SPIKE_BUFFER_CU

#include "kernel.cu_h"
#include "incoming.cu_h"


__constant__ size_t c_incomingPitch; // word pitch


__host__
void
setIncomingPitch(size_t pitch)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_incomingPitch),
				&pitch, sizeof(size_t), 0, hipMemcpyHostToDevice));
}



/*! \return the buffer number to use for the given delay, given current cycle */
__device__
uint
incomingSlot(uint cycle, uint delay1)
{
	return (cycle + delay1) % MAX_DELAY;
}



/* Return offset into full buffer data structure to beginning of buffer for a
 * particular targetPartition and a particular delay. */
__device__
uint
incomingBufferStart(uint targetPartition, uint cycle, uint delay1)
{
	return (targetPartition * MAX_DELAY + incomingSlot(cycle, delay1)) * c_incomingPitch;
}



__device__
incoming_t
getIncoming(uint cycle, uint offset, incoming_t* g_incoming)
{
	return g_incoming[incomingBufferStart(CURRENT_PARTITION, cycle, 0) + offset];
}




/*! \return incoming spike group from a particular source */
__device__
incoming_t
make_incoming(uint sourcePartition, uint sourceNeuron, uint delay)
{
	ASSERT(sourcePartition < (1<<PARTITION_BITS));
	ASSERT(sourceNeuron < (1<<NEURON_BITS));
	ASSERT(delay < (1<<DELAY_BITS));

	return (incoming_t) { sourcePartition, sourceNeuron, delay };
}


__device__
uint
incomingDelay(incoming_t in)
{
	return in.delay;
}


__device__
uint
incomingPartition(incoming_t in)
{
	return in.source_partition;
}



__device__
uint
incomingNeuron(incoming_t in)
{
	return in.source_neuron;
}



/*! \return address into matrix with number of incoming synapse groups */
__device__
size_t
incomingCountAddr(uint targetPartition, uint cycle, uint delay1)
{
	return targetPartition * MAX_DELAY + incomingSlot(cycle, delay1);
}


#endif
