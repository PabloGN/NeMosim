#include "hip/hip_runtime.h"
/*! \todo could use a constant+shared memory lookup table for this instead of
 * computing exponentials */

#include "log.hpp"

/* STDP parameters
 *
 * The STDP parameters apply to all neurons in a network. One might, however,
 * want to do this differently for different neuron populations. This is not
 * yet supported.
 *
 * The STDP parameters are stored in constant memory as we're running out of
 * available kernel paramters.
 *
 * We postfix parameters either P or D to indicate whether the parameter refers
 * to potentiation or depression.
 *
 * - tau specifies the maximum delay between presynaptic spike and
 *   postsynaptic firing for which STDP has an effect.
 * - alpha is a multiplier for the exponential
 */

__constant__ int c_stdpTauP;
__constant__ int c_stdpTauD;
__constant__ float c_stdpTauInvP;
__constant__ float c_stdpTauInvD;
__constant__ float c_stdpAlphaP;
__constant__ float c_stdpAlphaD;


#define SET_STDP_PARAMETER(symbol, val) CUDA_SAFE_CALL(\
        hipMemcpyToSymbol(HIP_SYMBOL(symbol), &val, sizeof(val), 0, hipMemcpyHostToDevice)\
    )

__host__
void
configureStdp(int tauP, int tauD, float alphaP, float alphaD)
{
    float tauInvP = (float) (1.0 / (double) tauP);
    float tauInvD = (float) (1.0 / (double) tauD);
    SET_STDP_PARAMETER(c_stdpTauP, tauP);
    SET_STDP_PARAMETER(c_stdpTauD, tauD);
    SET_STDP_PARAMETER(c_stdpTauInvP, tauInvP);
    SET_STDP_PARAMETER(c_stdpTauInvD, tauInvD);
    SET_STDP_PARAMETER(c_stdpAlphaP, alphaP);
    SET_STDP_PARAMETER(c_stdpAlphaD, alphaD);
}


/* In the kernel we load the parameters into shared memory. These variables can
 * then be accessed using broadcast */

__shared__ int s_stdpTauP;
__shared__ int s_stdpTauD;
__shared__ float s_stdpTauInvP;
__shared__ float s_stdpTauInvD;
__shared__ float s_stdpAlphaP;
__shared__ float s_stdpAlphaD;


#define LOAD_STDP_PARAMETER(symbol) s_ ## symbol = c_ ## symbol

__device__
void
loadStdpParameters()
{
    //! \todo could use an array for this and load in parallel
    if(threadIdx.x == 0) {
        LOAD_STDP_PARAMETER(stdpTauP);
        LOAD_STDP_PARAMETER(stdpTauD);
        LOAD_STDP_PARAMETER(stdpTauInvP);
        LOAD_STDP_PARAMETER(stdpTauInvD);
        LOAD_STDP_PARAMETER(stdpAlphaP);
        LOAD_STDP_PARAMETER(stdpAlphaD);
    }
    __syncthreads();
}



__device__
float
depression(int dt)
{
	return s_stdpAlphaD * exp(__int2float_rn(-dt)*s_stdpTauInvD);
}


__device__
float
potentiation(int dt)
{
	return s_stdpAlphaP * exp(__int2float_rn(-dt)*s_stdpTauInvP);
}


/*! Process each firing neuron, potentiating synapses with spikes reaching the
 * fired neuron shortly before firing. */
__device__
void
updateLTP(
	uint maxDelay,
	uint currentTime,
	uint s_maxL0SynapsesR,
	// reverse connectivity
	uint* g_cmR, size_t cmPitchR, size_t cmSizeR,
	// forward connectivity
	uint* g_cmF, size_t cmPitchF, size_t cmSizeF,
	uint16_t* s_firingIdx,
	//! \todo change to uint?
	int s_firingCount,
	uint32_t* s_recentArrivals,
	uint32_t* g_arrivalDelays)
{
	/*! \note This is the maximum number of chunks required for this whole
	 * cluster. It should be possible to reduce this for rows with few
	 * entries. Perhaps better to just save the number of chunks in
	 * constant memory. It would depend on the chunk size, though. */
	//! \todo change to uint
	__shared__ int s_chunkCount;
	__shared__ int s_synapsesPerDelay;
	__shared__ int s_delaysPerChunk;
	__shared__ int s_chunksPerDelay;

	float* g_ltp = (float*) (g_cmR + RCM_LTP * cmSizeR);

	//! \todo factor this out and share with integrate step
	if(threadIdx.x == 0) {
		//! \todo do we need to round to block size if multiple chunks per delay?
		s_synapsesPerDelay = ALIGN(s_maxL0SynapsesR, warpSize);
		s_chunksPerDelay = DIV_CEIL(s_synapsesPerDelay, THREADS_PER_BLOCK);
		s_delaysPerChunk = THREADS_PER_BLOCK / s_synapsesPerDelay;
	}
	__syncthreads();

	for(int i=0; i<s_firingCount; ++i) {

		int postsynaptic = s_firingIdx[i];

		__shared__ uint s_delayBlocks;
		__shared__ uint32_t s_arrivals[MAX_DELAY];

		if(s_recentArrivals[postsynaptic]) {

			//! \todo factor this out and share with integrate step
			if(threadIdx.x == 0) {
				s_delayBlocks = 0;

				/* It's probably not worthwhile pre-loading arrival delays, since
				 * only a few of the loaded values will be used */
				//! \todo could pre-load in one go for all the ones that did fire, though
				uint32_t arrivalBits = g_arrivalDelays[postsynaptic];
				while(arrivalBits) {
					int arrivalDelay = __ffs(arrivalBits) - 1;
					s_arrivals[s_delayBlocks] = arrivalDelay;
					arrivalBits &= ~(0x1 << arrivalDelay);
					s_delayBlocks += 1;
				}
				s_chunkCount = s_delaysPerChunk == 0 ?
					s_delayBlocks * s_chunksPerDelay :  // >= 1 chunk(s) per delay
					DIV_CEIL(s_delayBlocks, s_delaysPerChunk);  // multiple delays per chunk
			}
			__syncthreads();

			for(int chunk=0; chunk < s_chunkCount; ++chunk) {

				int delayEntry = s_delaysPerChunk == 0 ?
					chunk / s_chunksPerDelay :
					chunk * s_delaysPerChunk + threadIdx.x / s_synapsesPerDelay;
				uint32_t delay = s_arrivals[delayEntry];
				/* Offset /within/ a delay block */
				int synapseIdxR = s_delaysPerChunk == 0 ?
					(chunk % s_chunksPerDelay) * THREADS_PER_BLOCK + threadIdx.x :
					(threadIdx.x % s_synapsesPerDelay);

				// reverse matrix *only* contains excitatory neurons
				//! \todo consider using per-neuron maximum here instead
				if(synapseIdxR < s_maxL0SynapsesR 
						&& delayEntry < s_delayBlocks
#ifdef __DEVICE_EMULATION__
						// warp size is 1, so rounding to warp size not as expected
						&& threadIdx.x < s_synapsesPerDelay * s_delaysPerChunk
#endif
					)
				{

					size_t synapseAddressR = 
						postsynaptic * maxDelay * cmPitchR
						+ delay * cmPitchR
						+ synapseIdxR;

					uint sdataR = g_cmR[synapseAddressR];

					if(sdataR != INVALID_REVERSE_SYNAPSE) {

						/* The delivery time of the last spike on this synapse is
						 * recorded in the forward matrix. */
						uint synapseIdxF = forwardIdx(sdataR);

						size_t forwardAddress = 
							sourceNeuron(sdataR) * maxDelay * cmPitchF
							+ delay * cmPitchF
							+ synapseIdxF;

						uint sdataF = g_cmF[forwardAddress];

						int dt = currentTime - arrivalTime(sdataF);
						//assert(dt > 0);
						if(dt < s_stdpTauP) {
							g_ltp[synapseAddressR] += potentiation(dt);
							DEBUG_MSG("ltp +%f for synapse %u -> %u after delay of %u\n",
									potentiation(dt), sourceNeuron(sdataR), postsynaptic, dt);
						}
					}
				}
			}
			__syncthreads();
		}
        __syncthreads();
	}
}




//! \todo factor out and share with deliverL0
__device__
void
setDelayBits(
	// input
	uint32_t delayBits,
	// output
	uint* s_delayBlocks,
	uint32_t* s_delays
)
{
	if(threadIdx.x == 0) {
		uint delayBlocks = 0;
		while(delayBits) {
			int arrivalDelay = __ffs(delayBits) - 1;
			s_delays[delayBlocks] = arrivalDelay;
			delayBits &= ~(0x1 << arrivalDelay);
			delayBlocks += 1;
		}
#if 0
		s_chunkCount = s_delaysPerChunk == 0 ?
			delayBlocks * s_chunksPerDelay :  // >= 1 chunk(s) per delay
			DIV_CEIL(delayBlocks, s_delaysPerChunk);  // multiple delays per chunk
#endif
		*s_delayBlocks = delayBlocks;
	}
	__syncthreads();
}



__device__
void
setPartitionParameters(uint* s_partitionSize, uint* s_neuronsPerThread)
{
    if(threadIdx.x == 0) {
        *s_partitionSize = c_partitionSize[CURRENT_PARTITION];
        *s_neuronsPerThread = DIV_CEIL(*s_partitionSize, THREADS_PER_BLOCK);
	}
	__syncthreads();
}



__global__
void
clearSTDPAccumulator_(
		uint maxPartitionSize,
		uint maxDelay,
		// Delay bits
		uint32_t* g_delayBits,
		size_t pitch32,
		// Accumulator
		float* g_acc,		//! \note caller should point to correct part of multi-dimensional matrix
		size_t pitch,
		size_t size)
{
	//! \todo add timing of this kernel as well

	__shared__ uint s_partitionSize;
	__shared__ uint s_neuronsPerThread;

	setPartitionParameters(&s_partitionSize, &s_neuronsPerThread);

	/* Pre-load all delay bits, since all of it will be needed */
	__shared__ uint32_t s_delayBits[MAX_PARTITION_SIZE];
	STDP_FN(loadSharedArray)(s_partitionSize, s_neuronsPerThread, pitch32, g_delayBits, s_delayBits);

	//! \todo time this without loading delay bits 
	for(uint presynaptic=0; presynaptic<s_partitionSize; ++presynaptic) {

		__shared__ uint s_delayBlocks;
		__shared__ uint32_t s_delays[MAX_DELAY];

		setDelayBits(s_delayBits[presynaptic], &s_delayBlocks, s_delays);

		ASSERT(pitch <= THREADS_PER_BLOCK);

		//! \todo deal with several delays in parallel as in L0 delivery (see also addL0LTD)
		//! \todo deal with multiple chunks per delay
		for(uint delayIdx=0; delayIdx<s_delayBlocks; ++delayIdx) {
			uint delay = s_delays[delayIdx];
			//! \todo make this work even if there are more threads than delays
			if(threadIdx.x < pitch) {
				size_t g_offset = (presynaptic * maxDelay + delay) * pitch + threadIdx.x;
				g_acc[g_offset] = 0;
			}
		}
	}
}


/* Re-order long-term potentiation from the reverse order (by postsynaptic)
 * used in the accumulation array, to the forward order (by presynaptic) used
 * in the synaptic weight matrix. 
 *
 * prefix r: reverse matrix
 * prefix f: forward matrix
 */
__global__
void
reorderLTP_(
#ifdef KERNEL_TIMING
	unsigned long long* g_cc,
	size_t ccPitch,
#endif
	int maxPartitionSize,
	int maxDelay,
	size_t pitch32,
	uint32_t* g_delayBits,
	// forward connectivity
	uint* gf_cm,
	size_t f_pitch,
	size_t f_size,
	// reverse connectivity
	uint* gr_cm,
	size_t r_pitch,
	size_t r_size)
{
	SET_COUNTER(s_ccReorderSTDP, 0);

	/* The accumulated long-term potentiation is stored in a reverse-order matrix. */
	__shared__ int s_partitionSize;
	__shared__ int s_neuronsPerThread;
	if(threadIdx.x == 0) {
		s_partitionSize = c_partitionSize[CURRENT_PARTITION];
		s_neuronsPerThread = DIV_CEIL(s_partitionSize, THREADS_PER_BLOCK);
	}
	__syncthreads();

	/* Pre-load all delay bits, since all of it will be needed */
	__shared__ uint32_t s_delayBits[MAX_PARTITION_SIZE];
	STDP_FN(loadSharedArray)(s_partitionSize, s_neuronsPerThread, pitch32, g_delayBits, s_delayBits);

	size_t poffset = CURRENT_PARTITION * maxPartitionSize * maxDelay;
	uint* g_raddress =       gr_cm + RCM_ADDRESS * r_size + poffset * r_pitch;
	float* gr_ltp = (float*) gr_cm + RCM_LTP     * r_size + poffset * r_pitch;
	float* gf_ltp = (float*) gf_cm + CM_FLTP     * f_size + poffset * f_pitch;

	for(uint postsynaptic=0; postsynaptic < s_partitionSize; ++postsynaptic) {

		__shared__ uint s_delayBlocks;
		__shared__ uint32_t s_delays[MAX_DELAY];
		setDelayBits(s_delayBits[postsynaptic], &s_delayBlocks, s_delays);

		ASSERT(r_pitch <= THREADS_PER_BLOCK);

		for(int delayIdx=0; delayIdx<s_delayBlocks; ++delayIdx) {

			int delay = s_delays[delayIdx];
			//! \todo make this work even if there are more threads than delays
			if(threadIdx.x < r_pitch) {
				size_t gr_offset = (postsynaptic * maxDelay + delay) * r_pitch + threadIdx.x;
				uint rsynapse = g_raddress[gr_offset];
				if(rsynapse != INVALID_REVERSE_SYNAPSE) {

					float ltp = gr_ltp[gr_offset];

					if(ltp != 0.0f) {

						size_t gf_offset 
								= (sourceNeuron(rsynapse) * maxDelay + delay) * f_pitch 
								+ forwardIdx(rsynapse);

						gf_ltp[gf_offset] = ltp;
						gr_ltp[gr_offset] = 0;

						DEBUG_MSG("stdp %+f for synapse %u -> %u\n",
							ltp, sourceNeuron(rsynapse), postsynaptic);
					}
				}
			}
		}
	}

	SET_COUNTER(s_ccReorderSTDP, 1);
	WRITE_COUNTERS(s_ccReorderSTDP, g_cc, ccPitch, 2);
}




/*! Apply STDP, i.e. modify synapses using the accumulated LTP and LTD statistics, 
 * modulated by reward. Synapse weights are limited to [0, maxWeight]. Synapses
 * which are already 0, are not potentiated */
__global__
void
applySTDP_(
#ifdef KERNEL_TIMING
	unsigned long long* g_cc,
	size_t ccPitch,
#endif
	float reward,
	float maxWeight,
	int maxPartitionSize, // not warp aligned
	int maxDelay,
	size_t pitch32,
	uint32_t* g_delayBits,
	uint* g_cm,
	size_t pitch,
	size_t size,
	bool recordTrace)
{
	SET_COUNTER(s_ccApplySTDP, 0);

	__shared__ uint s_partitionSize;
	__shared__ uint s_neuronsPerThread;
	setPartitionParameters(&s_partitionSize, &s_neuronsPerThread);

	/* Pre-load all delay bits, since all of it will be needed */
	__shared__ uint32_t s_delayBits[MAX_PARTITION_SIZE];
	STDP_FN(loadSharedArray)(s_partitionSize, s_neuronsPerThread, pitch32, g_delayBits, s_delayBits);

	size_t partitionOffset = CURRENT_PARTITION * maxPartitionSize * maxDelay * pitch;
#ifdef __DEVICE_EMULATION__
	uint* g_postsynaptic =      g_cm + CM_ADDRESS * size + partitionOffset;
#endif
	float* g_weights = (float*) g_cm + CM_WEIGHT     * size + partitionOffset;
	float* g_ltp     = (float*) g_cm + CM_FLTP       * size + partitionOffset;
	float* g_ltd     = (float*) g_cm + CM_LTD        * size + partitionOffset;
	uint* g_trace    =          g_cm + CM_STDP_TRACE * size + partitionOffset;

	for(uint presynaptic=0; presynaptic<s_partitionSize; ++presynaptic) {

		__shared__ uint s_delayBlocks;
		__shared__ uint32_t s_delays[MAX_DELAY];

		setDelayBits(s_delayBits[presynaptic], &s_delayBlocks, s_delays);
		ASSERT(pitch <= THREADS_PER_BLOCK);

		//! \todo deal with several delays in parallel as in L0 delivery
		//! \todo deal with multiple chunks per delay
		for(uint delayIdx=0; delayIdx<s_delayBlocks; ++delayIdx) {
			uint delay = s_delays[delayIdx];

			//! \todo make this work even if there are more threads than delays
			if(threadIdx.x < pitch) {

				size_t g_offset 
					= (presynaptic * maxDelay + delay) * pitch 
					+ threadIdx.x;

				float ltp = g_ltp[g_offset];
				float ltd = g_ltd[g_offset];
				float w_diff = reward * (ltp + ltd);

				if(w_diff != 0.0f) {

					float w_old = g_weights[g_offset];
					float w_new = fmin(maxWeight, fmax(w_old + w_diff, 0.0f));

					/* Only modify excitatory synapses. Also, don't modify
					 * weight once it has reached 0. */
					//! \todo for synapses with zero weight, don't write to accumulator in the first place
					if(w_old > 0.0f && w_old != w_new) {
						g_weights[g_offset] = w_new;
						DEBUG_MSG("stdp, updated synapse %u -> %u to %f\n",
							presynaptic, targetNeuron(g_postsynaptic[g_offset]), w_new);

						if(recordTrace) {
							g_trace[g_offset] = __float_as_int(w_new);
						}
					}
				}

				if(ltp != 0.0f) {
					g_ltp[g_offset] = 0.0f;
				}

				if(ltd != 0.0f) {
					g_ltd[g_offset] = 0.0f;
				}

			}
		}
	}
	SET_COUNTER(s_ccApplySTDP, 1);
	WRITE_COUNTERS(s_ccApplySTDP, g_cc, ccPitch, 2);
}
