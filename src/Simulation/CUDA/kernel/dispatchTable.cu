#include "hip/hip_runtime.h"
#ifndef DISPATCH_TABLE_CU
#define DISPATCH_TABLE_CU

#include "kernel.cu_h"
#include "dispatchTable.cu_h"


/*! \return
 *		Word pitch of connectivity matrix block
 */
__device__
size_t
f0_pitch(fcm_ref_t r)
{
	return (size_t) r.x;
}



__device__
uint32_t*
f0_base(fcm_ref_t r)
{
#ifdef __DEVICE_EMULATION__
	uint64_t ptr = r.z;
	ptr <<= 32;
	ptr |= r.y;
	return (uint32_t*) ptr;
#else
	return (uint32_t*) r.y;
#endif
}



/*! \return	size (in words) of one plane of the given connectivity matrix */
__device__
size_t
f0_size(fcm_ref_t r)
{
	return MAX_PARTITION_SIZE * f0_pitch(r);
}



/*!
 * \param ref
 *		Reference to connectivity matrix block for a particular partition/delay
 * \return
 * 		Address of the beginning of the addressing part of connectivity matrix
 * 		block specified by \a ref
 */
__device__
uint*
f0_address(fcm_ref_t ref)
{
	return f0_base(ref) + FCM_ADDRESS * MAX_PARTITION_SIZE * f0_pitch(ref);
}



//! \todo rename
/* \return
 * 		Address of the beginning of the addressing part of forward connectivity
 * 		matrix block.
 */
__device__
uint*
f0_address2(uint32_t* base, size_t pitch)
{
	return base + FCM_ADDRESS * MAX_PARTITION_SIZE * pitch;
}



/*!
 * \param ref
 *		Reference to connectivity matrix block for a particular partition/delay
 * \return
 * 		Address of the beginning of the weights part of connectivity matrix
 * 		block specified by \a ref
 */
__device__
float*
f0_weights(fcm_ref_t ref)
{
	return (float*) f0_base(ref) + FCM_WEIGHT * MAX_PARTITION_SIZE * f0_pitch(ref);
}



/* \return
 * 		Address of the beginning of the weights part of forward connectivity
 * 		matrix block.
 */
__device__
float*
f0_weights2(uint32_t* base, size_t pitch)
{
	return (float*) base + FCM_WEIGHT * MAX_PARTITION_SIZE * pitch;
}



__host__
fcm_ref_t
fcm_packReference(void* address, size_t pitch)
{
	assert(sizeof(address) <= sizeof(uint64_t));

	uint64_t ptr64 = (uint64_t) address;

#ifdef __DEVICE_EMULATION__
	uint32_t low = (uint32_t) (ptr64 & 0xffffffff);
	uint32_t high = (uint32_t) ((ptr64 >> 32) & 0xffffffff);
	return make_uint4((uint) pitch, (uint) low, (uint) high, 0);
#else
	const uint64_t MAX_ADDRESS = 4294967296LL; // on device
	assert(ptr64 < MAX_ADDRESS);
	return make_uint2((uint) pitch, (uint) ptr64);
#endif
}


//! \todo rename once we have removed old format
texture<fcm_ref_t, 3, hipReadModeElementType> tf1_refs2;



__device__
fcm_ref_t
getFCM2(uint sourcePartition, uint targetPartition, uint delay0)
{
	return tex3D(tf1_refs2, (float) delay0, (float) targetPartition, (float) sourcePartition);
}



__host__
hipArray*
f1_setDispatchTable2(
		size_t partitionCount,
		size_t delayCount,
		const std::vector<fcm_ref_t>& h_table)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<fcm_ref_t>();

	hipArray* d_table;
	hipExtent ext = make_hipExtent(delayCount, partitionCount, partitionCount);
	CUDA_SAFE_CALL(hipMalloc3DArray(&d_table, &channelDesc, ext));

	hipMemcpy3DParms copyParams = {0};
	copyParams.extent = ext;
	copyParams.kind = hipMemcpyHostToDevice;
	copyParams.dstArray = d_table;
	copyParams.srcPtr = make_hipPitchedPtr(
			(void*)&h_table[0],
			ext.width * sizeof(fcm_ref_t), ext.width, ext.height);
	CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));

	// set texture parameters
	tf1_refs2.addressMode[0] = hipAddressModeClamp;
	tf1_refs2.addressMode[1] = hipAddressModeClamp;
	tf1_refs2.addressMode[2] = hipAddressModeClamp;
	tf1_refs2.filterMode = hipFilterModePoint;
	tf1_refs2.normalized = false;
	CUDA_SAFE_CALL(hipBindTextureToArray(tf1_refs2, d_table, channelDesc));
	return d_table;
}



/* For L0 delivery, we load for all delays  */
//! \todo rename after cleanup
__device__
void
loadDispatchTable2_L0_(uint32_t* s_fcmAddr[], ushort2 s_fcmPitch[])
{
	if(threadIdx.x < MAX_DELAY) {
		fcm_ref_t fcm = getFCM2(CURRENT_PARTITION, CURRENT_PARTITION, threadIdx.x);
		s_fcmAddr[threadIdx.x] = f0_base(fcm);
		s_fcmPitch[threadIdx.x].x = f0_pitch(fcm);
		s_fcmPitch[threadIdx.x].y = DIV_CEIL(f0_pitch(fcm), THREADS_PER_BLOCK);
	}
	__syncthreads();
}

#endif
