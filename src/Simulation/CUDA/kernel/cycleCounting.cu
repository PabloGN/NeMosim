#include "hip/hip_runtime.h"
#include "cycleCounting.cu_h"

/* Each kernel has a separate cycle counter */
#ifdef KERNEL_TIMING
__shared__ clock_t s_ccMain[CC_MAIN_COUNT];
//! \todo don't allocate this memory if STDP not enabled
__shared__ clock_t s_ccReorderSTDP[CC_STDP_REORDER_COUNT];
__shared__ clock_t s_ccApplySTDP[CC_STDP_APPLY_COUNT];
#endif


/* Calculate the duration based on start and end times. The clock counters are
 * 32-bit and silently wrap around, at least on G80. It seems that clock_t is
 * not neccessarily 32-bit, so we need a bit of hard-coding here. This is not
 * future-proof, obviously. */
__device__
clock_t
duration(clock_t start, clock_t end)
{
	if (end > start)
		return end - start;
	else
		return end + (0xffffffff - start);
}

__device__
void
setCycleCounter(clock_t* s_cc, size_t counter)
{
	if(threadIdx.x == 0) {
		s_cc[counter] = clock();
	}
}



__device__
void
writeCycleCounters(clock_t* s_cc, unsigned long long* g_cc, size_t pitch, size_t count)
{
    __syncthreads();
	if(threadIdx.x < count-1) {
		clock_t d = duration(s_cc[threadIdx.x], s_cc[threadIdx.x+1]);
		atomicAdd(g_cc + blockIdx.x * pitch + threadIdx.x, (unsigned long long) d);
	}
}


#ifdef KERNEL_TIMING
//! \todo add separate methods for start and end counters?
#define SET_COUNTER(s_cc, counter) setCycleCounter(s_cc, counter)
#define WRITE_COUNTERS(s_cc, g_cc, ccPitch, ccCount) writeCycleCounters(s_cc, g_cc, ccPitch, ccCount)
#else
#define SET_COUNTER(s_cc, counter)
#define WRITE_COUNTERS(s_cc, g_cc, ccPitch, ccCount)
#endif
