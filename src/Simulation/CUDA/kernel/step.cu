#include "hip/hip_runtime.h"
#include "cycle.cu"

#undef STDP_FN
#ifdef STDP
#define STDP_FN(f) f ## _STDP
#else
#define STDP_FN(f) f ## _static
#endif


/*! Combined integrate and fire using sparse connectivity matrix, a single step
* updates the state (u and v) of each neuron and produces spikes to be used in
* the next simulation cycle. 
* 
* The number of neurons per block provided to the kernel is always
* warp-aligned. This means that some threads do useless work, but at no cost.
* Using a warp-aligned neuron number simplifies the control when the number of
* neurons is not an exact multiple of the number of threads per block.
*
 * The parameters (a, b, c, and d) can be set for each individual neuron and
 * should be pre-loaded in global memory before this kernel is invoked.
 */
__global__
void
STDP_FN(step) (
		int substeps,
		uint32_t cycle,
		uint64_t* g_recentFiring,
		// neuron state
		float* g_neuronParameters,
		unsigned* g_rngState,
		//! \todo combine with g_neuronParameters
		float* g_sigma,
		size_t neuronParametersSize,
		// connectivity
		uint* gf0_cm, uint64_t* gf0_delays,
		uint* gf1_cm, uint64_t* gf1_delays,
		// L1 spike queue
		uint2* gSpikeQueue,
		size_t sqPitch,
		unsigned int* gSpikeQueueHeads,
		size_t sqHeadPitch,
		// firing stimulus
		uint32_t* g_fstim,
		size_t pitch1,
#ifdef KERNEL_TIMING
		// cycle counting
		unsigned long long* g_cycleCounters,
		size_t ccPitch,
#endif
		uint32_t* firingOutput) // already offset to current cycle
{
	SET_COUNTER(s_ccMain, 0);

	/* The shared memory is allocated in fixed-sized blocks. During the
	 * different stages of the kernel each block may be used for different
	 * purposes. */

	/* Per-neuron buffers */
	__shared__ uint32_t s_M1KA[MAX_PARTITION_SIZE];
	__shared__ uint64_t s_M1KB[MAX_PARTITION_SIZE];

	/* Per-thread buffers */
	__shared__ uint16_t s_T16[THREADS_PER_BLOCK];
	__shared__ uint32_t s_T32[THREADS_PER_BLOCK];

	/* Per-delay buffer */
	__shared__ uint32_t s_D32[MAX_DELAY];

	/* Per-partition buffer */
	__shared__ uint32_t s_P32[MAX_PARTITION_COUNT];

	uint64_t* s_recentFiring = s_M1KB;

	/* Per-partition parameters */
	__shared__ uint s_partitionSize;
	__shared__ uint sf1_maxSynapsesPerDelay;
	__shared__ float s_substepMult;

	__shared__ uint32_t* s_fcmAddr[MAX_DELAY];
	__shared__ ushort2 s_fcmPitch[MAX_DELAY]; // ... and pre-computed chunk count

	if(threadIdx.x == 0) {
#ifdef __DEVICE_EMULATION__
		s_cycle = cycle;
#endif
		s_partitionSize = c_partitionSize[CURRENT_PARTITION];
		sf1_maxSynapsesPerDelay = cf1_maxSynapsesPerDelay[CURRENT_PARTITION];
		s_substepMult = 1.0f / __int2float_rn(substeps);
    }
	__syncthreads();

	loadNetworkParameters();

#ifdef STDP
	loadStdpParameters();
#endif
	/* Within a connection matrix plane, partitionRow is the row offset of the
	 * current partition. The offset in /words/ differ between forward/reverse
	 * and level 0/1 as they have different row pitches */
	size_t f_partitionRow = CURRENT_PARTITION * s_maxPartitionSize * s_maxDelay;

	SET_COUNTER(s_ccMain, 1);

    //! \todo no need to clear array here, if loading thalamic input
	setSharedArray(s_M1KA, 0);
	float* s_current = (float*) s_M1KA;
    if(g_rngState != NULL && g_sigma != NULL) {
        thalamicInput(s_partitionSize,
                neuronParametersSize,
                s_pitch32,
                g_rngState,
                g_sigma,
                s_current);
    }

	SET_COUNTER(s_ccMain, 2);

	loadSharedArray(s_partitionSize,
			s_pitch64,
			g_recentFiring + readBuffer(cycle) * PARTITION_COUNT * s_pitch64,
			s_recentFiring);
	__syncthreads();

	SET_COUNTER(s_ccMain, 3);

	bool haveL1 = gSpikeQueue != NULL;
	if(haveL1) {
		STDP_FN(gatherL1Spikes_JIT_)(
				readBuffer(cycle),
				gSpikeQueue,
				sqPitch,
				gSpikeQueueHeads,
				sqHeadPitch,
				s_current,
				s_P32);
	}

	SET_COUNTER(s_ccMain, 4);

	deliverL0Spikes_(
			s_partitionSize,
			s_recentFiring,
			gf0_delays + CURRENT_PARTITION * s_pitch64,
			s_current, s_T16, s_T32, s_D32,
			s_fcmAddr, s_fcmPitch);

	SET_COUNTER(s_ccMain, 5);

	/* The dense firing output is staged in shared memory before being written
	 * to global memory */
	clearFiringOutput();

	//__shared__ uint32_t s_fstim[DIV_CEIL(STDP_FN(MAX_PARTITION_SIZE), 32)];
	//! \todo use the same buffer for both input and output
	/* Make sure s_T16 is large enough */
	uint32_t* s_fstim = (uint32_t*) s_T16;
	bool hasExternalInput = g_fstim != 0;
	ASSERT(THREADS_PER_BLOCK/2 >= DIV_CEIL(MAX_PARTITION_SIZE, 32));
	loadExternalFiring(hasExternalInput, s_partitionSize, pitch1, g_fstim, s_fstim);

	fire(
			s_partitionSize,
			substeps, s_substepMult,
			pitch1,
			g_neuronParameters + CURRENT_PARTITION * s_pitch32,
			neuronParametersSize,
			s_current, 
			s_fstim);

	__syncthreads();

	writeFiringOutput(firingOutput + CURRENT_PARTITION * pitch1, pitch1);

	SET_COUNTER(s_ccMain, 6);

#ifdef STDP
	updateSTDP_(
			false,
			s_recentFiring,
			s_recentFiring,
			s_pitch32,
			s_partitionSize,
			cr0_address, cr0_stdp, cr0_pitch,
			s_T32);
#endif
	SET_COUNTER(s_ccMain, 7);
#ifdef STDP
	if(haveL1) {
		updateSTDP_(
				true,
				g_recentFiring + readBuffer(cycle) * PARTITION_COUNT * s_pitch64,
				s_recentFiring,
				s_pitch64,
				s_partitionSize,
				cr1_address, cr1_stdp, cr1_pitch,
				s_T32);
	}
#endif
	SET_COUNTER(s_ccMain, 8);

	/* We need the (updated) recent firing history for L1 spike
	 * delivery later, but won't update this further, so we can write
	 * back to global memory now. */
	updateHistory(s_partitionSize, s_recentFiring,
			g_recentFiring
				+ writeBuffer(cycle) * PARTITION_COUNT * s_pitch64
				+ CURRENT_PARTITION * s_pitch64);
	//! \todo add an additional counter?

	if(haveL1) {
		STDP_FN(deliverL1Spikes_JIT)(
				s_maxDelay,
                writeBuffer(cycle),
				s_partitionSize,
				//! \todo need to call this differently from wrapper
				sf1_maxSynapsesPerDelay,
				gf1_cm + f_partitionRow * sf1_pitch, sf1_pitch, sf1_size,
				s_recentFiring,
				gf1_delays + CURRENT_PARTITION * s_pitch64,
				(uint2*) s_M1KA, // used for s_current previously, now use for staging outgoing spikes
				//! \todo compile-time assertions to make sure we're not overflowing here
				//! \todo fix naming!
				gSpikeQueue,
				sqPitch,
				gSpikeQueueHeads,
				sqHeadPitch,
				s_T16, s_T32, s_D32, s_P32,
				s_fcmAddr, s_fcmPitch);
	}

	SET_COUNTER(s_ccMain, 9);
	WRITE_COUNTERS(s_ccMain, g_cycleCounters, ccPitch, CC_MAIN_COUNT);
}
