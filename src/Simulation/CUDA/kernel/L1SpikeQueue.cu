#include "hip/hip_runtime.h"
#include "kernel.cu_h"
#include "util.h"
#include <stdint.h>
#ifdef __DEVICE_EMULATION__
#   include <assert.h>
#   include <stdio.h>
#endif


/* The current cycle indicates which half of the double buffer is for reading
 * and which is for writing */
__device__
uint
readBuffer(uint cycle)
{
    return (cycle & 0x1) ^ 0x1;
}


__device__
uint
writeBuffer(uint cycle)
{
    return cycle & 0x1;
}


/*! \return word offset to beginning of spike buffer for a particular partition
 * pair */
__device__
size_t
sbBase(size_t pitch, size_t src, size_t tgt, size_t bufferIdx)
{
	ASSERT(src < PARTITION_COUNT);
	ASSERT(tgt < PARTITION_COUNT);
	ASSERT(bufferIdx <= 1);
	return ((tgt * PARTITION_COUNT + src) * 2 + bufferIdx) * pitch;
}



//! \todo fix these constants

#define BUFFER_SZ 16
//! \todo modify L1 delivery to handle more buffers than threads
//#define BUFFER_COUNT MAX_PARTITION_COUNT
#define BUFFER_COUNT THREADS_PER_BLOCK



/*! Each possible source buffer has a counter (the 'buffer head') specifying
 * how many spikes are due for delivery. The buffer heads is a 2D matrix with
 * one row for each target partition
 *
 * The buffer head matrix is written to during spike scatter and read from
 * during spike gather. Since the accesses in these two cases are in different
 * order, so one access will necessarily be non-coalesced. It does not matter
 * which it is.
 *
 * \todo it would be possible to set the row pitch such that the non-coalesced
 * access avoids bank conflicts. It would still be non-coalesced, though.
 */


/*! \return the offset into the global memory buffer head matrix for a
 * particular partition pair */
__device__
size_t
headOffset(size_t src, size_t tgt, size_t pitch, size_t bufferIdx)
{
	ASSERT(src < PARTITION_COUNT);
	ASSERT(tgt < PARTITION_COUNT);
	ASSERT(bufferIdx <= 1);
    return bufferIdx * PARTITION_COUNT * pitch + tgt * pitch + src;
}



/*! Load and clear all the buffer heads when processing incoming spikes in the
 * target partition. The buffer heads need to be cleared so that the buffer is
 * ready to be filled again. Any synapses in the buffer is left there as
 * garbage. */
__device__
void
loadAndClearBufferHeads(
        uint32_t* g_heads,
        uint32_t* s_heads,
        size_t pitch,
        size_t bufferIdx)
{
#if MAX_THREAD_BLOCKS > THREADS_PER_BLOCK
#error	"Need to rewrite loadL1Current to load spike queue heads in several loads"
#endif
	int sourcePartition = threadIdx.x;
	if(sourcePartition < PARTITION_COUNT) {
		size_t offset = headOffset(sourcePartition, CURRENT_PARTITION, pitch, bufferIdx);
		//! \todo could use atomicExch here instead. Not sure which is faster.
		s_heads[sourcePartition] = g_heads[offset];
		g_heads[offset] = 0;
	}
	__syncthreads();
}




/* Flush spike buffer (up to maxSlot) for a single partition.
 *
 * \todo write 4B values instead of 8B
 */
__device__
void
flushSpikeBuffer(
	uint writeBufferIdx,
	uint count,
	int targetPartition,
	uint* s_heads,
	uint2* s_outbuf64,
	uint2* g_sq64,
	size_t sqPitch)
{
	/* We only have one warp's worth of data here. To improve bandwidth
	 * utilisation write 4B per thread rather than 8B. */
	// uint* s_outbuf32 = (uint*) s_outbuf64;
	// uint* g_sq32 = (uint*) g_sq64;

	if(threadIdx.x < count) {
		//! \todo simplify addressing once old L1CM is removed
		size_t base = sbBase(sqPitch, CURRENT_PARTITION, targetPartition, writeBufferIdx);
		//uint data = s_outbuf32[targetPartition * BUFFER_SZ * 2 + threadIdx.x];
		uint2 data = s_outbuf64[targetPartition * BUFFER_SZ + threadIdx.x];
		//g_sq32[2 * (base + s_heads[targetPartition]) + threadIdx.x] = data;
		g_sq64[base + s_heads[targetPartition] + threadIdx.x] = data;
		DEBUG_MSG("Sending L1 current %f for synapse %d-?? -> %u-%u (after unknown delay)\n",
			__int_as_float(data.y), CURRENT_PARTITION,
			targetPartition, targetNeuron(data.x));
	}
}


/*! Flush all spike buffers */
__device__
void
flushAllSpikeBuffers(
	uint writeBufferIdx,
	size_t headPitch,
	uint32_t* g_heads,
	uint32_t* s_heads,
	uint* s_outheads,
	uint2* s_outbuf,
	uint2* g_sq,
	size_t sqPitch)
{
	/* Determine global buffer offsets in parallel, and at the same time flush
	 * the buffer head to global memory. The head buffer is repurposed to now
	 * contain the offset into the buffer entry */
	uint targetPartition = threadIdx.x;
	if(targetPartition < PARTITION_COUNT) {
		size_t offset = headOffset(CURRENT_PARTITION, targetPartition, headPitch, writeBufferIdx);
		g_heads[offset] = s_heads[targetPartition] + s_outheads[targetPartition];
	}
	__syncthreads();

	//! \todo factor out function to flush all
	/* Now flush all buffers which still have data in them */
	for(int targetPartition=0; targetPartition<PARTITION_COUNT; ++targetPartition) {
		//! \todo could load the sizes in parallel here, without using atomics
		flushSpikeBuffer(
            writeBufferIdx,
			s_outheads[targetPartition],
			targetPartition,
			s_heads,
			s_outbuf,
			g_sq, sqPitch);
	}
}



/*! Update current buffer with incoming spikes */
__device__
void
updateCurrent(
		uint readBufferIdx, // double buffer index
		uint sourcePartition,
		uint32_t spikeIdx,   // index of spike for current thread
		uint32_t spikeCount, // number of spikes to delivered for current buffer 
		uint2* g_sq,
		size_t sqPitch,
		float* s_current)
{
#define BANKS 16
	//! \todo remove hard-coding
	//! \todo do this for all threads
	__shared__ uint s_committing[BANKS];
	if(threadIdx.x < BANKS) {
		s_committing[threadIdx.x] = 0;
	}
	__syncthreads();

	//! \todo conditional load from global memory

	uint targetPartition = CURRENT_PARTITION;
	size_t base = sbBase(sqPitch, sourcePartition, targetPartition, readBufferIdx);
	//! \todo load by 32-bit values
	uint2 spike = g_sq[base + spikeIdx];

	/* We don't need to clear the data from the spike buffer,
	 * as long as the head is cleared. \see loadAndClearBufferHeads */
	float weight = __int_as_float(spike.y);
	uint target = targetNeuron(spike.x);

	bool spiked = weight != 0.0f && spikeIdx < spikeCount;

	uint commitNo;
	if(spiked) {
		// serialise commits for each shared memory bank to avoid race condition
		commitNo = atomicAdd(s_committing + (target % BANKS), 1);
	}
	__syncthreads();

	/* In the worst case *every* spike has the same target. Determine the
	 * maximum number of threads that need to be serialised. */
	//! \todo use reduction to find the maximum here
	__shared__ uint s_maxCommit;
	if(threadIdx.x == 0) {
		s_maxCommit = 0;
		for(uint i=0; i<BANKS; ++i) {
			s_maxCommit = max(s_maxCommit, s_committing[i]);
		}
	}
	__syncthreads();

	for(uint commit=0; commit <= s_maxCommit; ++commit) {
		if(spiked && commitNo == commit) {
			s_current[targetNeuron(spike.x)] += weight;
			ASSERT(targetNeuron(spike.x) < MAX_PARTITION_SIZE);
			DEBUG_MSG("Receiving L1 current %f from %d-?? to %d-%d\n",
					weight, sourcePartition, targetPartition, targetNeuron(spike.x));
		}
		__syncthreads();
	}
	__syncthreads();
}



/*! Load all incoming spikes from L1 connetivity into current accumulator */
__device__
void
gatherL1Spikes_JIT_(
		uint readBufferIdx,
		uint2* g_sq,
		size_t sqPitch,
		uint* g_heads,
        size_t headPitch,
		float* s_current,
        uint32_t* s_heads)
{
	loadAndClearBufferHeads(g_heads, s_heads, headPitch, readBufferIdx);
	for(uint src=0; src<PARTITION_COUNT; ++src) {
		uint parallelLoads = DIV_CEIL(s_heads[src], THREADS_PER_BLOCK);
		for(uint load=0; load<parallelLoads; ++load) {
			uint spikeIdx = load * THREADS_PER_BLOCK + threadIdx.x;
			updateCurrent(readBufferIdx, src, spikeIdx, s_heads[src],
					g_sq, sqPitch, s_current);
		}
	}
	__syncthreads();
}


/* TODO: the loop structure here is nearly the same as deliverL0Spikes. Factor
 * out or use a code generator to avoid repetition */
__device__
void
deliverL1Spikes_JIT(
	uint maxDelay,
	uint writeBufferIdx,
	uint partitionSize,
	uint sf1_maxSynapses,
	uint* gf1_cm, uint f1_pitch, uint f1_size,
	uint32_t* s_recentFiring,
	//! \todo STDP support
#ifdef STDP
	//uint32_t* s_recentIncoming,
	//float* g_ltd,
	//uint stdpCycle,
#endif
	uint32_t* g_firingDelays,
	// L1 spike queue
    //! \todo allow more than 32 partitions (by splitting L1CM)
    uint2* s_outbuf,        // 16 words of buffer per target partition
	uint2* g_sq,
	size_t sqPitch,
	uint* g_heads,
    size_t headPitch)
{
	uint*  gf1_address =          gf1_cm + FCM_ADDRESS * f1_size;
	float* gf1_weights = (float*) gf1_cm + FCM_WEIGHT  * f1_size;

	/*! \note This is the maximum number of chunks required for this whole
	 * cluster. It should be possible to reduce this for rows with few entries.
	 * Perhaps better to just save the number of chunks in constant memory. It
	 * would depend on the chunk size, though. */
	__shared__ uint s_chunkCount;

	/* L1 spikes are delivered via a global memory buffer. Writes to these
	 * buffers may be quite scattered. To reduce the impact of non-coalesced
	 * writes we therefore stage synaptic data in shared memory before writing
	 * it to global memory.
	 *
	 * We therefore have two buffer heads to keep track of: one for the global
	 * per target-partition spike buffer (which may be filled from several
     * presynaptic neurons here), and one for the local per target-partition
     * buffer. The latter is the 'outbuffer', while the former is just
     * 'buffer'.  */

	/*! \todo use one of the general-purpose chunks of shared memory */
	/* Can't pack these into one array of uint2's, since we need to do atomic
	 * operations */
	__shared__ uint s_heads[BUFFER_COUNT];
	__shared__ uint s_outheads[BUFFER_COUNT];
	//! \todo factor out method here, using a template function
	for(int i=0; i < BUFFER_COUNT/THREADS_PER_BLOCK; ++i) {
		s_heads[i*THREADS_PER_BLOCK + threadIdx.x] = 0;
		s_outheads[i*THREADS_PER_BLOCK + threadIdx.x] = 0;
	}

	__shared__ uint s_synapsesPerDelay;
	__shared__ uint s_chunksPerDelay;
	__shared__ uint s_delaysPerChunk;
	if(threadIdx.x == 0) {
		//! \todo do we need to round to block size if multiple chunks per delay?
		s_synapsesPerDelay = ALIGN(sf1_maxSynapses, warpSize);
		s_chunksPerDelay = DIV_CEIL(s_synapsesPerDelay, THREADS_PER_BLOCK);
		s_delaysPerChunk = THREADS_PER_BLOCK / s_synapsesPerDelay;
	}
	__syncthreads();


	for(int preOffset=0; preOffset < partitionSize; preOffset += THREADS_PER_BLOCK) {

		__shared__ uint s_firingCount;
		//! \todo make this a re-usable chunk of memory
		__shared__ uint16_t s_firingIdx[THREADS_PER_BLOCK];
		__shared__ uint32_t s_arrivalBits[THREADS_PER_BLOCK];

		if(threadIdx.x == 0) {
			s_firingCount = 0;
		}
		__syncthreads();

		//! \todo load s_recentFiring here, write result to smem array
		int candidate = preOffset + threadIdx.x;
		uint32_t arrivals = s_recentFiring[candidate] & g_firingDelays[candidate];
		if(arrivals && candidate < partitionSize) {
			int nextFree = atomicAdd(&s_firingCount, 1);
			s_firingIdx[nextFree] = candidate;
			s_arrivalBits[nextFree] = arrivals;
		}
		__syncthreads();

		/* We now have the indices of the firing of THREADS_PER_BLOCK
		 * presynaptic neurons */
		for(int i=0; i<s_firingCount; ++i) {

			int presynaptic = s_firingIdx[i];

			__shared__ uint s_delayBlocks;
			__shared__ uint32_t s_arrivals[MAX_DELAY];
			if(threadIdx.x == 0) {
				s_delayBlocks = 0;
				uint32_t arrivalBits = s_arrivalBits[i];

				while(arrivalBits) {
					int arrivalDelay = __ffs(arrivalBits) - 1;
					s_arrivals[s_delayBlocks] = arrivalDelay;
					arrivalBits &= ~(0x1 << arrivalDelay);
					s_delayBlocks += 1;
				}
				s_chunkCount = s_delaysPerChunk == 0 ?
					s_delayBlocks * s_chunksPerDelay :  // >= 1 chunk(s) per delay
					DIV_CEIL(s_delayBlocks, s_delaysPerChunk);  // multiple delays per chunk
			}
			__syncthreads();

			/* The delay pitch may vary between networks, or between partitions.
			 * Even with sequential processing of presynaptic neurons, we want to
			 * maximise parallel processing of incoming spikes from different
			 * delays. We have two situations:
			 *
			 * 1) if the delay pitch is more than half the block size we process
			 *    each delay sequentially
			 * 2) if the delay pitch is less than or equal to half the block size
			 *    we process multiple delays in parallel
			 */

			for(int chunk=0; chunk < s_chunkCount; ++chunk) {

				int delayEntry = s_delaysPerChunk == 0 ?
					chunk / s_chunksPerDelay :
					threadIdx.x / s_synapsesPerDelay;
				uint32_t delay = s_arrivals[delayEntry];
				/* Offset /within/ a delay block */
				int synapseIdx = s_delaysPerChunk == 0 ?
					(chunk % s_chunksPerDelay) * THREADS_PER_BLOCK + threadIdx.x :
					(threadIdx.x % s_synapsesPerDelay);

				float weight;
				uint target;
				int bufferIdx = 0; // relative to beginning of output buffer slot
				bool doCommit = false;

				//! \todo consider using per-neuron maximum here instead
				if(synapseIdx < sf1_maxSynapses && delayEntry < s_delayBlocks
#ifdef __DEVICE_EMULATION__
						// warp size is 1, so rounding to warp size not as expected
						&& threadIdx.x < s_synapsesPerDelay * s_delaysPerChunk
#endif
				  ) {
					size_t synapseAddress =
						(presynaptic * maxDelay + delay) * f1_pitch + synapseIdx;
					weight = gf1_weights[synapseAddress];
					target = gf1_address[synapseAddress];

					if(weight != 0.0f) {
						doCommit = true;
						bufferIdx = atomicAdd(s_outheads + targetPartition(target), 1);
						//! \todo deal with STDP here
					}
				}

				/* For L1 delivery there's no race condition in the scatter
				 * step, but if care is not taken here, we get one in the
				 * gather step, as multiple spikes may converge on the same
				 * postsynaptic neuron at the same time.
				 *
				 * While we don't need to worry about race conditions, we _do_
				 * need to worry about memory bandwidth. A single firing neuron
				 * can generate spikes reaching many different targets, spread
				 * over multiple target partitions. If we naïvely deal with one
				 * firing neuron at a time and write the spike to the global
				 * memory spike queue directly, we end up with a large number
				 * of non-coalesced writes.
				 *
				 * To reduce this problem, we buffer outgoing data on a per
				 * target-partition basis. The buffers are kept as small as is
				 * reasonable (one warp) and then flushed as needed. */

				/* In the worst case, every thread writes to the same target
				 * partition, which means the buffer will easily overflow. We
				 * therefore need to interleave the filling of the output
				 * buffer with its flushing */

				//! \todo could use warp vote here, if some care is taken
				/* The number of buffers is exactly the warp size, so we
				 * can use a single warp vote to determine if we need to
				 * flush anything */
				//! \todo make this compile-time assertion
				//! \todo factor out
				__shared__ uint s_flushCount;
				__shared__ uint s_flushPartition[BUFFER_COUNT];
				do {
					/* ensure loop condition is not changed while threads are
					 * in different loop iterations */
					__syncthreads();

					/* Write one batch of data to output buffers, up to the
					 * limit of the buffer */
					if(doCommit && bufferIdx < BUFFER_SZ) {
						//! \todo do some compression here to avoid race conditions later
						s_outbuf[targetPartition(target) * BUFFER_SZ + bufferIdx] =
							make_uint2(target, __float_as_int(weight));
						doCommit = false;
						DEBUG_MSG("Buffering L1 current %f for synapse"
								"%u-?? -> %u-%u (after unknown delay)\n",
								weight, CURRENT_PARTITION,
								targetPartition(target), targetNeuron(target));
					} else {
						bufferIdx -= BUFFER_SZ; // prepare to write to buffer on subsequent loop iteration
					}

					/* Determine how many buffers are now full and need flushing */
					if(threadIdx.x == 0) {
						s_flushCount = 0;
					}
					__syncthreads();

					//! \todo factor out
					{
						ASSERT(BUFFER_COUNT <= THREADS_PER_BLOCK);
						int targetPartition = threadIdx.x;
						if(targetPartition < BUFFER_COUNT) {
							if(s_outheads[targetPartition] >= BUFFER_SZ) {
								s_outheads[targetPartition] -= BUFFER_SZ;
								uint next = atomicInc(&s_flushCount, BUFFER_COUNT);
								s_flushPartition[next] = targetPartition;
							}
						}
						__syncthreads();
					}

					/* Flush buffers */
					/*! \todo could potentially flush multiple buffers in one go here */
					for(int flush_i=0; flush_i < s_flushCount; ++flush_i) {
						int targetPartition = s_flushPartition[flush_i];
						DEBUG_THREAD_MSG(0, "Flushing buffer %d\n", targetPartition);
						flushSpikeBuffer(
                                writeBufferIdx,
								BUFFER_SZ, // flush whole buffer
								targetPartition,
								s_heads,
								s_outbuf,
								g_sq, sqPitch);
						__syncthreads();
						//! \todo could add all in parallel?
						if(threadIdx.x == 0) {
							s_heads[targetPartition] += BUFFER_SZ;
						}
					}
				} while(s_flushCount);
				__syncthreads(); // ensure every thread has left the loop, before re-entering it
			}
		}
	}

	flushAllSpikeBuffers(writeBufferIdx, headPitch, g_heads, s_heads, s_outheads, s_outbuf, g_sq, sqPitch);
	DEBUG_MSG("End deliver L1\n");
}

