#include "hip/hip_runtime.h"
#ifndef NEMO_CUDA_CURRENT_CU
#define NEMO_CUDA_CURRENT_CU

/*! \file current.cu Functions related to neuron input current */

/* Copyright 2010 Imperial College London
 *
 * This file is part of NeMo.
 *
 * This software is licenced for non-commercial academic use under the GNU
 * General Public Licence (GPL). You should have received a copy of this
 * licence along with nemo. If not, see <http://www.gnu.org/licenses/>.
 */

#include "fixedpoint.cu"


/*! \brief Add input current for a particular neuron
 *
 * The input current is stored in shared memory in a fixed-point format. This
 * necessitates overflow detection, so that we can use saturating arithmetic.
 *
 * \param[in] neuron
 *		0-based index of the target neuron
 * \param[in] current
 *		current in mA in fixed-point format
 * \param s_currentE
 *		shared memory vector containing current from EPSPs for all neurons in partition
 * \param s_currentI
 *		shared memory vector containing current from IPSPs for all neurons in partition
 * \param[out] s_overflow
 *		bit vector indicating overflow status for all neurons in partition
 * \param[out] s_negative
 *		bit vector indicating the overflow sign for all neurons in partition
 *
 * \pre neuron < partition size
 * \pre all shared memory buffers have at least as many entries as partition size
 *
 * \todo add cross-reference to fixed-point format documentation
 */
__device__
void
addCurrent(nidx_t neuron, fix_t current, fix_t s_current[], uint32_t s_overflow[])
{
	ASSERT(neuron < MAX_PARTITION_SIZE);
	bool overflow = fx_atomicAdd(s_current + neuron, current);
	bv_atomicSetPredicated(overflow, neuron, s_overflow);
#ifndef FIXPOINT_SATURATION
	ASSERT(!overflow);
#endif
}



/*! \brief Add externally provided current stimulus
 *
 * The user can provide per-neuron current stimulus
 * (nemo::cuda::Simulation::addCurrentStimulus).
 *
 * \param[in] partition
 *		\i global index of partition
 * \param[in] psize
 *		number of neurons in current partition
 * \param[in] pitch
 *		pitch of g_current, i.e. distance in words between each partitions data
 * \param[in] g_current
 *		global memory vector containing current for all neurons in partition.
 *		If set to NULL, no input current will be delivered.
 * \param s_current
 *		shared memory vector containing current for all neurons in partition
 *
 * \pre neuron < size of current partition
 * \pre all shared memory buffers have at least as many entries as the size of
 * 		the current partition
 *
 * \see nemo::cuda::Simulation::addCurrentStimulus
 */
__device__
void
addCurrentStimulus(
		unsigned partition,
		unsigned psize,
		size_t pitch,
		const float* g_current,
		float* s_current)
{
	if(g_current != NULL) {
		for(unsigned nbase=0; nbase < psize; nbase += THREADS_PER_BLOCK) {
			unsigned neuron = nbase + threadIdx.x;
			unsigned pstart = partition * pitch;
			float stimulus = g_current[pstart + neuron];
			s_current[neuron] += stimulus;
#ifdef NEMO_CUDA_PLUGIN_DEBUG_TRACE
			DEBUG_MSG_SYNAPSE("c%u %u-%u: +%f (external)\n",
					s_cycle, partition, neuron, g_current[pstart + neuron]);
#endif
		}
		__syncthreads();
	}
}



/*! Copy per-neuron accumulated current between two memory areas
 *
 * \param[in] current_in per-neuron accumulated current (shared or global memory)
 * \param[out] current_out per-neuron accumulated current (shared or global memory)
 *
 * Global memory arguments must be offset to the appropriate partition.
 */
__device__
void
copyCurrent(unsigned nNeurons, float* current_in, float* current_out)
{
	for(unsigned bNeuron=0; bNeuron < nNeurons; bNeuron += THREADS_PER_BLOCK) {
		unsigned neuron = bNeuron + threadIdx.x;
		current_out[neuron] = current_in[neuron];
	}
}




/*
 * \param pcount
 *		partition count considering \em all neuron types. Note that for some
 *		kernels PARTITION_COUNT refers to this global count, whereas for other
 *		kernels it refers to a local count.
 * \param partition
 *		\em global index of current partition
 *
 * \return gmem pointer to accumulated incoming excitatory current
 */
__device__
float*
incomingExcitatory(float* g_base, unsigned /* pcount */, unsigned partition, size_t pitch32)
{
	return g_base + partition * pitch32;
}



/*
 * \param pcount
 *		partition count considering \em all neuron types. Note that for some
 *		kernels PARTITION_COUNT refers to this global count, whereas for other
 *		kernels it refers to a local count.
 * \param partition
 *		\em global index of current partition

 * \return gmem pointer to accumulated incoming inhbitory current
 */
__device__
float*
incomingInhibitory(float* g_base, unsigned pcount, unsigned partition, size_t pitch32)
{
	return g_base + (pcount + partition) * pitch32;
}

#endif
