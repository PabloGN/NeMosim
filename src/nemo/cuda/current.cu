#include "hip/hip_runtime.h"
#ifndef NEMO_CUDA_CURRENT_CU
#define NEMO_CUDA_CURRENT_CU

/*! \file current.cu Functions related to neuron input current */

/* Copyright 2010 Imperial College London
 *
 * This file is part of NeMo.
 *
 * This software is licenced for non-commercial academic use under the GNU
 * General Public Licence (GPL). You should have received a copy of this
 * licence along with nemo. If not, see <http://www.gnu.org/licenses/>.
 */


/*! \brief Load externally provided current stimulus from gmem
 *
 * The user can provide per-neuron current stimulus
 * (nemo::cuda::Simulation::addCurrentStimulus).
 *
 * \param[in] partition
 *		\i global index of partition
 * \param[in] psize
 *		number of neurons in current partition
 * \param[in] pitch
 *		pitch of g_current, i.e. distance in words between each partitions data
 * \param[in] g_current
 *		global memory vector containing current for all neurons in partition.
 *		If set to NULL, no input current will be delivered.
 * \param[out] s_current
 *		shared memory vector which will be set to contain input stimulus (or
 *		zero, if there's no stimulus).
 *
 * \pre neuron < size of current partition
 * \pre all shared memory buffers have at least as many entries as the size of
 * 		the current partition
 *
 * \see nemo::cuda::Simulation::addCurrentStimulus
 */
__device__
void
loadCurrentStimulus(
		unsigned partition,
		unsigned psize,
		size_t pitch,
		const float* g_current,
		float* s_current)
{
	if(g_current != NULL) {
		for(unsigned nbase=0; nbase < psize; nbase += THREADS_PER_BLOCK) {
			unsigned neuron = nbase + threadIdx.x;
			unsigned pstart = partition * pitch;
			float stimulus = g_current[pstart + neuron];
			s_current[neuron] = stimulus;
#ifdef NEMO_CUDA_PLUGIN_DEBUG_TRACE
			DEBUG_MSG_SYNAPSE("c%u %u-%u: +%f (external)\n",
					s_cycle, partition, neuron, g_current[pstart + neuron]);
#endif
		}
	} else {
		for(unsigned nbase=0; nbase < psize; nbase += THREADS_PER_BLOCK) {
			unsigned neuron = nbase + threadIdx.x;
			s_current[neuron] = 0;
		}
	}
	__syncthreads();
}



/* \return address for a given current accumulator for a given partition
 *
 * \param pcount
 *		total number of partitions in the network
 * \param partition
 *		index of partition of interest
 * \param accIndex
 *		accumulator index. Accumulators are indexed from 0, and there's
 *		typically one accumulator per synapse type.
 */
__device__
float*
accumulator(float* g_base, unsigned pcount, unsigned partition, unsigned accIndex, size_t pitch32)
{
	return g_base + (accIndex * pcount + partition) * pitch32;
}



#endif
