#ifndef CONNECTIVITY_MATRIX_CU
#define CONNECTIVITY_MATRIX_CU

/* Copyright 2010 Imperial College London
 *
 * This file is part of nemo.
 *
 * This software is licenced for non-commercial academic use under the GNU
 * General Public Licence (GPL). You should have received a copy of this
 * licence along with nemo. If not, see <http://www.gnu.org/licenses/>.
 */

#include <assert.h>

#include <nemo/util.h>

#include "kernel.cu_h"
#include "connectivityMatrix.cu_h"

#define NEURON_MASK MASK(NEURON_BITS)
#define PARTITION_MASK MASK(PARTITION_BITS)
#define DELAY_MASK MASK(DELAY_BITS)

#define PARTITION_SHIFT NEURON_BITS

/* Reverse synapses */
#define R_FSYNAPSE_SHIFT (R_PARTITION_SHIFT + PARTITION_BITS)
#define R_PARTITION_SHIFT (R_NEURON_SHIFT + NEURON_BITS)
#define R_NEURON_SHIFT DELAY_BITS



/* distance (in words) between a synapses's address data and its weight data. */
__constant__ size_t c_fcmPlaneSize;

__host__
synapse_t
f_nullSynapse()
{
	return 0;
}


__host__
hipError_t
setFcmPlaneSize(size_t sz)
{
	return hipMemcpyToSymbol(HIP_SYMBOL(c_fcmPlaneSize),
				&sz, sizeof(size_t), 0, hipMemcpyHostToDevice);
}



__host__ __device__
unsigned
targetNeuron(unsigned synapse)
{
#ifdef __DEVICE_EMULATION__
    return synapse & NEURON_MASK;
#else
	return synapse;
#endif
}


__host__
unsigned
r_packSynapse(unsigned sourcePartition, unsigned sourceNeuron, unsigned delay)
{
	assert(!(sourcePartition & ~PARTITION_MASK));
	assert(!(sourceNeuron & ~NEURON_MASK));
	assert(!(delay & ~DELAY_MASK));
	return (sourcePartition << R_PARTITION_SHIFT)
	     | (sourceNeuron    << R_NEURON_SHIFT)
	     |  delay;
}



__device__ __host__
unsigned
sourceNeuron(unsigned rsynapse)
{
    return (rsynapse >> R_NEURON_SHIFT) & NEURON_MASK;
}


__device__ __host__
unsigned
sourcePartition(unsigned rsynapse)
{
    return (rsynapse >> R_PARTITION_SHIFT) & PARTITION_MASK;
}



__device__ __host__
unsigned
r_delay1(unsigned rsynapse)
{
    return rsynapse & DELAY_MASK; 
}


__device__
unsigned
r_delay0(unsigned rsynapse)
{
	return r_delay1(rsynapse) - 1;
}



/* To improve packing of data in the connectivity matrix, we use different
 * pitches for each partition */
__constant__ size_t cr_pitch[MAX_PARTITION_COUNT];

/* We also need to store the start of each partitions reverse connectivity
 * data, to support fast lookup. This data should nearly always be in the
 * constant cache */
__constant__ uint32_t* cr_address[MAX_PARTITION_COUNT];

/* Ditto for the STDP accumulators */
__constant__ weight_dt* cr_stdp[MAX_PARTITION_COUNT];

/* Ditto for the forward synapse offset */
__constant__ uint32_t* cr_faddress[MAX_PARTITION_COUNT];



#define SET_CR_ADDRESS_VECTOR(symbol, vec, len, type)                         \
    err = hipMemcpyToSymbol(HIP_SYMBOL(symbol), vec,                                     \
            len * sizeof(type), 0, hipMemcpyHostToDevice);                   \
    if(hipSuccess != err) {                                                  \
        return err;                                                           \
    }



__host__
hipError_t
configureReverseAddressing(
        size_t* r_pitch,
        uint32_t* const* r_address,
        weight_dt* const* r_stdp,
        uint32_t* const* r_faddress,
		size_t len)
{
	hipError_t err;
	//! \todo extend vectors and fill with NULLs
	SET_CR_ADDRESS_VECTOR(cr_pitch, r_pitch, len, size_t);
	SET_CR_ADDRESS_VECTOR(cr_address, r_address, len, uint32_t*);
	SET_CR_ADDRESS_VECTOR(cr_stdp, r_stdp, len, weight_dt*);
	SET_CR_ADDRESS_VECTOR(cr_faddress, r_faddress, len, uint32_t*);
	return hipSuccess;
}

#endif
