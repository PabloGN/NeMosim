#include "hip/hip_runtime.h"
/* Copyright 2010 Imperial College London
 *
 * This file is part of nemo.
 *
 * This software is licenced for non-commercial academic use under the GNU
 * General Public Licence (GPL). You should have received a copy of this
 * licence along with nemo. If not, see <http://www.gnu.org/licenses/>.
 */

#include "kernel.cu_h"
#include "thalamicInput.cu_h"



__device__
unsigned
neuronLocalStateIndex(unsigned neuron, unsigned plane, size_t pitch)
{
    return (plane * PARTITION_COUNT + CURRENT_PARTITION) * pitch + neuron;
}



//! \todo use unsigned4 instead?
__device__ 
void 
rng_loadState(unsigned *rngState, const unsigned* g_rngState, unsigned neuron, size_t pitch)
{
	for(unsigned i=0; i<4; i++){
		rngState[i] = g_rngState[neuronLocalStateIndex(neuron, i, pitch)];
	}
}


__device__ 
void 
rng_saveState(const unsigned *rngState, unsigned *g_rngState, unsigned neuron, size_t pitch)
{
	for(unsigned i=0; i<4; i++){
		g_rngState[neuronLocalStateIndex(neuron, i, pitch)] = rngState[i];
	}
}


__device__ 
unsigned 
rng_genUniform(unsigned *rngState)
{
	unsigned t = (rngState[0]^(rngState[0]<<11));
	rngState[0] = rngState[1];
	rngState[1] = rngState[2];
	rngState[2] = rngState[3];
	rngState[3] = (rngState[3]^(rngState[3]>>19))^(t^(t>>8));
	return rngState[3];
}



/* For various reasons this generates a pair of samples for each call. If nesc.
 * then you can just stash one of them somewhere until the next time it is
 * needed or something.  */
__device__
float2
rng_genGaussian(unsigned* rngState)
{
	float a = rng_genUniform(rngState) * 1.4629180792671596810513378043098e-9f;
	float b = rng_genUniform(rngState) * 0.00000000023283064365386962890625f;
	float r = sqrtf(-2*logf(1-b));
	return make_float2(sinf(a)*r, cosf(a)*r);
}


__device__
void
thalamicInput(
		size_t partitionSize,
		size_t pitch,
		unsigned* g_rngState,
		float* g_nparam,
		float* s_current)
{
	unsigned rngState[4];

	float* g_sigma = g_nparam +
			+ PARAM_SIGMA * PARTITION_COUNT * pitch
			+ CURRENT_PARTITION * pitch;

	for(unsigned nbase=0; nbase < partitionSize; nbase += THREADS_PER_BLOCK) {

		unsigned neuron = nbase + threadIdx.x;

		/* Copy the input state from memory into our local state */
		rng_loadState(rngState, g_rngState, neuron, pitch);


		if(neuron < partitionSize) {

			//! \todo make use of  both randoms
			float2 r = rng_genGaussian(rngState);

			/*! \bug It seems that if r.x is very small the result of the
			 * multiplication is NaN (at least if sigma is 0, possibly in other
			 * cases as well). This issue seems unrelated to fusing
			 * multiply-add operations. Forcing separate multiplication does
			 * not fix. For instance with sigma=0 and r.x=0x00007f80
			 * (4.57384e-41) we get a NaN result. Could not reproduce in
			 * standalone test-case, however. */
			//! \todo consider using fixed-point arithmetic here as well.
			s_current[neuron] += r.x * g_sigma[neuron];
		}

		/* Copy the current RNG state back to memory (not strictly necessary, you
		 * can just generate a new random state every time if you want). */
		rng_saveState(rngState, g_rngState, neuron, pitch);

	}

	__syncthreads();
}
