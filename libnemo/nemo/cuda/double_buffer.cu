
#include <hip/hip_runtime.h>
/* Copyright 2010 Imperial College London
 *
 * This file is part of nemo.
 *
 * This software is licenced for non-commercial academic use under the GNU
 * General Public Licence (GPL). You should have received a copy of this
 * licence along with nemo. If not, see <http://www.gnu.org/licenses/>.
 */

/* Data structures which are used for communication between different
 * partitions, need to be double buffered so as to avoid race conditions.
 * These functions return the double buffer index (0 or 1) for the given cycle,
 * for either the read or write part of the buffer */

__device__
unsigned
readBuffer(unsigned cycle)
{
    return (cycle & 0x1) ^ 0x1;
}


__device__
unsigned
writeBuffer(unsigned cycle)
{
    return cycle & 0x1;
}
