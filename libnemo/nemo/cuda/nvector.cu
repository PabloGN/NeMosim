
#include <hip/hip_runtime.h>
/* Access functions for per-neuron data.
 *
 * See NVector.hpp/NVector.ipp for host-side functionality
 */

__constant__ size_t c_pitch32;
__constant__ size_t c_pitch64;


__host__
hipError_t
nv_setPitch32(size_t pitch32)
{
	return hipMemcpyToSymbol(HIP_SYMBOL(c_pitch32), &pitch32, sizeof(size_t), 0, hipMemcpyHostToDevice);
}


__host__
hipError_t
nv_setPitch64(size_t pitch64)
{
	return hipMemcpyToSymbol(HIP_SYMBOL(c_pitch64), &pitch64, sizeof(size_t), 0, hipMemcpyHostToDevice);
}





