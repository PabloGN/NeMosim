#include "hip/hip_runtime.h"
/*! \file gather.cu Gather kernel */

/* Copyright 2010 Imperial College London
 *
 * This file is part of NeMo.
 *
 * This software is licenced for non-commercial academic use under the GNU
 * General Public Licence (GPL). You should have received a copy of this
 * licence along with nemo. If not, see <http://www.gnu.org/licenses/>.
 */

#include "types.h"

#include "log.cu_h"

#include "bitvector.cu"
#include "connectivityMatrix.cu"
#include "double_buffer.cu"
#include "fixedpoint.cu"
#include "globalQueue.cu"
#include "nvector.cu"
#include "thalamicInput.cu"


/*! \brief Add input current for a particular neuron
 *
 * The input current is stored in shared memory in a fixed-point format. This
 * necessitates overflow detection, so that we can use saturating arithmetic.
 *
 * \param[in] neuron
 *		0-based index of the target neuron
 * \param[in] current
 *		current in mA in fixed-point format
 * \param s_current
 *		shared memory vector containing current for all neurons in partition
 * \param[out] s_overflow
 *		bit vector indicating overflow status for all neurons in partition
 * \param[out] s_negative
 *		bit vector indicating the overflow sign for all neurons in partition
 *
 * \pre neuron < partition size
 * \pre all shared memory buffers have at least as many entries as partition size
 *
 * \todo add cross-reference to fixed-point format documentation
 */
__device__
void
addCurrent(nidx_t neuron,
		fix_t current,
		fix_t* s_current,
		uint32_t* s_overflow,
		uint32_t* s_negative)
{
	bool overflow = fx_atomicAdd(s_current + neuron, current);
	bv_atomicSetPredicated(overflow, neuron, s_overflow);
	bv_atomicSetPredicated(overflow && fx_isNegative(current), neuron, s_negative);
#ifndef FIXPOINT_SATURATION
	ASSERT(!overflow);
#endif
}



/*! \brief Add externally provided current stimulus
 *
 * The user can provide per-neuron current stimulus
 * (\ref nemo::cuda::Simulation::addCurrentStimulus).
 *
 * \param[in] psize
 *		number of neurons in current partition
 * \param[in] pitch
 *		pitch of g_current, i.e. distance in words between each partitions data
 * \param[in] g_current
 *		global memory vector containing current for all neurons in partition.
 *		If set to NULL, no input current will be delivered.
 * \param s_current
 *		shared memory vector containing current for all neurons in partition
 * \param s_overflow
 *		bit vector indicating overflow status for all neurons in partition.
 *		Entries here may already be set and are simply OR-ed with any new entries.
 * \param s_negative
 *		bit vector indicating the overflow sign for all neurons in partition
 *		Entries here may already be set and are simply OR-ed with any new entries.
 *
 * \pre neuron < size of current partition
 * \pre all shared memory buffers have at least as many entries as the size of
 * 		the current partition
 *
 * \see nemo::cuda::Simulation::addCurrentStimulus
 */
__device__
void
addCurrentStimulus(unsigned psize,
		size_t pitch,
		const fix_t* g_current,
		fix_t* s_current,
		uint32_t* s_overflow,
		uint32_t* s_negative)
{
	if(g_current != NULL) {
		for(unsigned nbase=0; nbase < psize; nbase += THREADS_PER_BLOCK) {
			unsigned neuron = nbase + threadIdx.x;
			unsigned pstart = CURRENT_PARTITION * pitch;
			fix_t stimulus = g_current[pstart + neuron];
			addCurrent(neuron, stimulus, s_current, s_overflow, s_negative);
			DEBUG_MSG_SYNAPSE("c%u %u-%u: +%f (external)\n",
					s_cycle, CURRENT_PARTITION, neuron,
					fx_tofloat(g_current[pstart + neuron]));
		}
		__syncthreads();
	}
}



/*! Write all per-neuron accumulated current to global memory
 *
 * The global memory roundtrip is so that the accumulation and fire steps can
 * be done in separate kernel invocations.
 *
 * \param[in] s_current Per-neuron accumulated current
 * \param[out] g_current Per-neuron accumulated current (with correct partition offset in gmem)

 */
__device__
void
storeAccumulatedCurrent(unsigned nNeurons, float* s_current, float* g_current)
{
	for(unsigned bNeuron=0; bNeuron < nNeurons; bNeuron += THREADS_PER_BLOCK) {
		unsigned neuron = bNeuron + threadIdx.x;
		g_current[neuron] = s_current[neuron];
	}
}



/*! Gather incoming current from all spikes due for delivery \e now
 *
 * The whole spike delivery process is described in more detail in \ref
 * cuda_delivery and cuda_gather.
 *
 * \param[in] cycle
 * 		Current cycle
 * \param[in] g_fcm
 *		Forward connectivity matrix in global memory
 * \param[in] g_gqFill
 *		Fill rate for global queue
 * \param[in] g_gqData
 *		Pointer to full global memory double-buffered global queue
 * \param[out] s_current
 *		per-neuron vector with accumulated current in fixed point format.
 * \param[out] s_overflow
 *		bit vector indicating overflow status for all neurons in partition.
 * \param[out] s_negative
 *		bit vector indicating the overflow sign for all neurons in partition
 */
__device__
void
gather( unsigned cycle,
		synapse_t* g_fcm,
		gq_entry_t* g_gqData,
		unsigned* g_gqFill,
		float* s_current,
		uint32_t* s_overflow, // 1b per neuron overflow detection
		uint32_t* s_negative) // ditto
{
	//! \todo move init of current to here, so that we can ensure that it's zero
	/* Update incoming current in-place in fixed-point format */
	fix_t* s_fx_current = (fix_t*) s_current;
	__shared__ unsigned s_incomingCount;

	bv_clear(s_overflow);
	bv_clear(s_negative);

	if(threadIdx.x == 0) {
		//! \todo use atomicExch here instead
		size_t addr = gq_fillOffset(CURRENT_PARTITION, readBuffer(cycle));
		s_incomingCount = g_gqFill[addr];
		g_gqFill[addr] = 0;
	}
	__syncthreads();

	/*! \note Could use THREADS_PER_BLOCK here, but we're bit low on shared
	 * memory. */
#define GROUP_SIZE 128

	//! \todo could this smem be re-used?
	__shared__ synapse_t* s_warpAddress[GROUP_SIZE];

	//! \todo rename variables here
	for(unsigned groupBase = 0; groupBase < s_incomingCount; groupBase += GROUP_SIZE) {

		__shared__ unsigned s_groupSize;

		unsigned group = groupBase + threadIdx.x;

		if(threadIdx.x == 0) {
			s_groupSize =
				(group + GROUP_SIZE) > s_incomingCount
				? s_incomingCount % GROUP_SIZE
				: GROUP_SIZE;
			DEBUG_MSG_SYNAPSE("c%u: group size=%u, incoming=%u\n", cycle, s_groupSize, s_incomingCount);
		}
		__syncthreads();

		if(threadIdx.x < s_groupSize) {
			gq_entry_t sgin = gq_read(readBuffer(cycle), group, g_gqData);
			s_warpAddress[threadIdx.x] = g_fcm + gq_warpOffset(sgin) * WARP_SIZE;
			DEBUG_MSG_SYNAPSE("c%u w%u -> p%u\n", cycle, gq_warpOffset(sgin), CURRENT_PARTITION);
		}

		__syncthreads();

		for(unsigned gwarp_base = 0; gwarp_base < s_groupSize; gwarp_base += WARPS_PER_BLOCK) {

			unsigned bwarp = threadIdx.x / WARP_SIZE; // warp index within a block
			unsigned gwarp = gwarp_base + bwarp;      // warp index within the global schedule

			unsigned postsynaptic;
			fix_t weight = 0;

			synapse_t* base = s_warpAddress[gwarp] + threadIdx.x % WARP_SIZE;

			/* only warps at the very end of the group are invalid here */
			if(gwarp < s_groupSize) {
				postsynaptic = targetNeuron(*base);
				weight = *((unsigned*)base + c_fcmPlaneSize);
			}

			if(weight != 0) {
				addCurrent(postsynaptic, weight, s_fx_current, s_overflow, s_negative);
				DEBUG_MSG_SYNAPSE("c%u p?n? -> p%un%u %+f [warp %u]\n",
						s_cycle, CURRENT_PARTITION, postsynaptic,
						fx_tofloat(weight), (s_warpAddress[gwarp] - g_fcm) / WARP_SIZE);
			}
		}
		__syncthreads(); // to avoid overwriting s_groupSize
	}
}



__global__
void
gather( bool thalamicInputEnabled,
		uint32_t cycle,
		// neuron state
		float* gf_neuronParameters,
		unsigned* gu_neuronState,
		// spike delivery
		synapse_t* g_fcm,
		gq_entry_t* g_gqData,      // pitch = c_gqPitch
		unsigned* g_gqFill,
		//! \todo just load this directly in the fire step
		fix_t* g_istim,
		float* g_current)
{
	__shared__ float s_current[MAX_PARTITION_SIZE];

	/* Per-neuron bit-vectors. See bitvector.cu for accessors */
	__shared__ uint32_t s_overflow[S_BV_PITCH];
	__shared__ uint32_t s_negative[S_BV_PITCH];

	/* Per-partition parameters */
	__shared__ unsigned s_partitionSize;

	if(threadIdx.x == 0) {
#ifdef NEMO_CUDA_DEBUG_TRACE
		s_cycle = cycle;
#endif
		s_partitionSize = c_partitionSize[CURRENT_PARTITION];
    }
	__syncthreads();

	for(int i=0; i<DIV_CEIL(MAX_PARTITION_SIZE, THREADS_PER_BLOCK); ++i) {
		s_current[i*THREADS_PER_BLOCK + threadIdx.x] = 0.0f;
	}

	gather(cycle, g_fcm, g_gqData, g_gqFill, s_current, s_overflow, s_negative);

	addCurrentStimulus(s_partitionSize, c_pitch32, g_istim, (fix_t*) s_current, s_overflow, s_negative);
	fx_arrSaturatedToFloat(s_overflow, s_negative, (fix_t*) s_current, s_current);

	/* Generating random input current really ought to be done /before/
	 * providing the input current (for better performance in MPI backend).
	 * However, we need to either provide fixed-point random input or do an
	 * additional conversion inside the thalamic input code in order for this
	 * to work. */
	if(thalamicInputEnabled) {
		thalamicInput(s_partitionSize, c_pitch32,
				gu_neuronState, gf_neuronParameters, s_current);
	}

	storeAccumulatedCurrent(s_partitionSize, s_current, g_current + CURRENT_PARTITION * c_pitch32);
}



__host__
hipError_t
gather( unsigned partitionCount,
		bool thalamicInputEnabled,
		unsigned cycle,
		float* df_neuronParameters,
		unsigned* du_neuronState,
		fix_t* d_istim,
		float* d_current,
		synapse_t* d_fcm,
		gq_entry_t* d_gqData,
		unsigned* d_gqFill)
{
	dim3 dimBlock(THREADS_PER_BLOCK);
	dim3 dimGrid(partitionCount);

	gather<<<dimGrid, dimBlock>>>(
			thalamicInputEnabled, cycle,
			// neuron data
			df_neuronParameters, du_neuronState,
			// spike delivery
			d_fcm, d_gqData, d_gqFill,
			d_istim,    // external input current
			d_current); // internal input current

	return hipGetLastError();
}
