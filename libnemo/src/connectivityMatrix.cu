#ifndef CONNECTIVITY_MATRIX_CU
#define CONNECTIVITY_MATRIX_CU

#include <assert.h>

#include "kernel.cu_h"
#include "connectivityMatrix.cu_h"
#include "util.h"

#define NEURON_MASK MASK(NEURON_BITS)
#define PARTITION_MASK MASK(PARTITION_BITS)
#define FSYNAPSE_MASK MASK(SYNAPSE_BITS)
#define DELAY_MASK MASK(DELAY_BITS)

#define PARTITION_SHIFT NEURON_BITS

/* Reverse synapses */
#define R_FSYNAPSE_SHIFT (R_PARTITION_SHIFT + PARTITION_BITS)
#define R_PARTITION_SHIFT (R_NEURON_SHIFT + NEURON_BITS)
#define R_NEURON_SHIFT DELAY_BITS



/* distance (in words) between a synapses's address data and its weight data. */
__constant__ size_t c_fcmPlaneSize;

__host__
synapse_t
f_nullSynapse()
{
	return 0;
}


__host__
void
setFcmPlaneSize(size_t sz)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_fcmPlaneSize),
				&sz, sizeof(size_t), 0, hipMemcpyHostToDevice));
}


__host__
uint
f_packSynapse(uint neuron)
{
	return neuron;
}


__host__ __device__
uint
targetNeuron(uint synapse)
{
#ifdef __DEVICE_EMULATION__
    return synapse & NEURON_MASK;
#else
	return synapse;
#endif
}


__host__
uint
r_packSynapse(
        uint sourcePartition,
        uint sourceNeuron,
        uint sourceSynapse,
        uint delay)
{
    assert(!(sourcePartition & ~PARTITION_MASK));
    assert(!(sourceNeuron & ~NEURON_MASK));
    assert(!(delay & ~DELAY_MASK));
    return (  (sourceSynapse & FSYNAPSE_MASK) << R_FSYNAPSE_SHIFT)
            | (sourcePartition                << R_PARTITION_SHIFT)
            | (sourceNeuron                   << R_NEURON_SHIFT)
            | delay;
}



__device__ __host__
uint
sourceNeuron(uint rsynapse)
{
    return (rsynapse >> R_NEURON_SHIFT) & NEURON_MASK;
}


__device__ __host__
uint
sourcePartition(uint rsynapse)
{
    return (rsynapse >> R_PARTITION_SHIFT) & PARTITION_MASK;
}


__device__ __host__
uint
forwardIdx(uint rsynapse)
{
    return rsynapse >> R_FSYNAPSE_SHIFT;
}



__device__ __host__
uint
r_delay1(uint rsynapse)
{
    return rsynapse & DELAY_MASK; 
}


__device__
uint
r_delay0(uint rsynapse)
{
	return r_delay1(rsynapse) - 1;
}



/* To improve packing of data in the connectivity matrix, we use different
 * pitches for each partition */
//! \todo store offset instead of pointers
__constant__ DEVICE_UINT_PTR_T cr_pitch[MAX_THREAD_BLOCKS];

/* We also need to store the start of each partitions reverse connectivity
 * data, to support fast lookup. This data should nearly always be in the
 * constant cache */
__constant__ DEVICE_UINT_PTR_T cr_address[MAX_THREAD_BLOCKS];

/* Ditto for the STDP accumulators */
__constant__ DEVICE_UINT_PTR_T cr_stdp[MAX_THREAD_BLOCKS];

/* Ditto for the forward synapse offset */
__constant__ DEVICE_UINT_PTR_T cr_faddress[MAX_THREAD_BLOCKS];


#define SET_CR_ADDRESS_VECTOR(symbol, vec) CUDA_SAFE_CALL(\
		hipMemcpyToSymbol(HIP_SYMBOL(symbol), &vec[0], vec.size() * sizeof(DEVICE_UINT_PTR_T), 0, hipMemcpyHostToDevice)\
	)




__host__
void
configureReverseAddressing(
        const std::vector<DEVICE_UINT_PTR_T>& r_pitch,
        const std::vector<DEVICE_UINT_PTR_T>& r_address,
        const std::vector<DEVICE_UINT_PTR_T>& r_stdp,
        const std::vector<DEVICE_UINT_PTR_T>& r_faddress)
{
	//! \todo extend vectors and fill with NULLs
	SET_CR_ADDRESS_VECTOR(cr_pitch, r_pitch);
	SET_CR_ADDRESS_VECTOR(cr_address, r_address);
	SET_CR_ADDRESS_VECTOR(cr_stdp, r_stdp);
	SET_CR_ADDRESS_VECTOR(cr_faddress, r_faddress);
}

#endif
