#include "hip/hip_runtime.h"

/*! \brief GPU/CUDA kernel for neural simulation using Izhikevich's model
 * 
 * The entry point for the kernel is 'step' which will do one or more
 * integrate-and-fire step.  
 *
 * \author Andreas Fidjeland
 */

#include <hip/device_functions.h>
#include <stdio.h>
#include <assert.h>

#include <STDP.hpp>

#include "util.h"
#include "time.hpp"
#include "error.cu"
#include "log.hpp"
#include "connectivityMatrix.cu"
#include "FiringOutput.hpp"
#include "RuntimeData.hpp"
#include "CycleCounters.hpp"
#include "partitionConfiguration.cu"
#include "cycleCounting.cu"
#include "ThalamicInput.hpp"
#include "applySTDP.cu"
#include "outgoing.cu"
#include "incoming.cu"

#include "thalamicInput.cu"
#include "kernel.cu"
#include "stdp.cu" // only used if STDP enabled
#include "step.cu"



__host__
void
applyStdp(
		unsigned long long* d_cc,
		size_t ccPitch,
		uint partitionCount,
		uint fractionalBits,
		synapse_t* d_fcm,
		const nemo::STDP<float>& stdpFn,
		float reward)
{
	dim3 dimBlock(THREADS_PER_BLOCK);
	dim3 dimGrid(partitionCount);

	applySTDP_<<<dimGrid, dimBlock>>>(
#ifdef KERNEL_TIMING
			d_cc, ccPitch,
#endif
			d_fcm,
			fixedPoint(reward, fractionalBits),
			fixedPoint(stdpFn.maxWeight(), fractionalBits),
			fixedPoint(stdpFn.minWeight(), fractionalBits));

	if(assertionsFailed(partitionCount, -1)) {
		clearAssertions();
	}
}



/*! Wrapper for the __global__ call that performs a single simulation step */
//! \todo don't return status_t here. Only deal with this in API layer
__host__
status_t
stepSimulation(RuntimeData* rtdata, uint32_t* d_fstim, uint32_t* d_fout)
{
	dim3 dimBlock(THREADS_PER_BLOCK);
	dim3 dimGrid(rtdata->partitionCount());

	//! \todo use cycle number from rtdata insteda
	static uint scycle = 0;
	DEBUG_MSG("cycle %u\n", scycle);
	scycle += 1;

	step<<<dimGrid, dimBlock>>>(
			rtdata->usingStdp(),
			rtdata->cycle(),
			rtdata->recentFiring->deviceData(),
			// neuron parameters
			rtdata->d_neurons(),
			rtdata->thalamicInput->deviceRngState(),
			rtdata->thalamicInput->deviceSigma(),
			rtdata->neuronVectorLength(),
			// spike delivery
			rtdata->cm()->d_fcm(),
			rtdata->cm()->outgoingCount(),
			rtdata->cm()->outgoing(),
			rtdata->cm()->incomingHeads(),
			rtdata->cm()->incoming(),
			// firing stimulus
			d_fstim,
			// cycle counting
#ifdef KERNEL_TIMING
			rtdata->cycleCounters->data(),
			rtdata->cycleCounters->pitch(),
#endif
			d_fout);

    if(assertionsFailed(rtdata->partitionCount(), scycle)) {
        fprintf(stderr, "checking assertions\n");
        clearAssertions();
        return KERNEL_ASSERTION_FAILURE;
    }

	hipError_t status = hipGetLastError();

	if(status != hipSuccess) {
		WARNING("c%u %s", rtdata->cycle(), hipGetErrorString(status));
		LOG("", "Kernel parameters: <<<%d, %d>>>\n",
			dimGrid.x, dimBlock.x);
		return KERNEL_INVOCATION_ERROR;
	}

	return KERNEL_OK;
}
